#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/partition.h>
#include <thrust/device_vector.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

// TODO: static variables for device memory, any extra info you need, etc
static bool sortByMaterial = false;
static bool hasGBuffer = false;
static bool useGBuffer = true;
static ShadeableIntersection* dev_intersections_gbuffer = NULL;
static thrust::device_vector<int> index_array;
static Mesh* dev_meshes = NULL;
static OctreeDev* dev_octrees = NULL;
static int meshSize = 0;
static int numTrees = 0;
static int nStreams = 4;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_intersections_gbuffer, pixelcount * sizeof(ShadeableIntersection));
	
	index_array.resize(pixelcount);
	thrust::sequence(thrust::device, index_array.begin(), index_array.end());

	meshSize = scene->meshes.size();
	hipMallocManaged(&dev_meshes, meshSize * sizeof(Mesh));
	hipMemcpy(dev_meshes, scene->meshes.data(), meshSize * sizeof(Mesh), hipMemcpyHostToDevice);
	for (int i = 0; i < meshSize; i++)
	{
		Mesh& mesh = dev_meshes[i];
		mesh.materialid = scene->meshes[i].materialid;
		mesh.numVertices = scene->meshes[i].numVertices;
		mesh.numIndices = scene->meshes[i].numIndices;
		mesh.boundingVolume = scene->meshes[i].boundingVolume;
		mesh.translation = scene->meshes[i].translation;
		mesh.rotation = scene->meshes[i].rotation;
		mesh.scale = scene->meshes[i].scale;
		mesh.transform = scene->meshes[i].transform;
		mesh.inverseTransform = scene->meshes[i].inverseTransform;
		mesh.invTranspose = scene->meshes[i].invTranspose;

		float* dev_vertices;
		unsigned short* dev_indices;
		hipMalloc(&dev_vertices, scene->meshes[i].numVertices * sizeof(float) * 3);
		hipMalloc(&dev_indices, scene->meshes[i].numIndices * sizeof(unsigned short));

		hipMemcpy(dev_vertices, scene->meshes[i].vertices, scene->meshes[i].numVertices * sizeof(float) * 3, hipMemcpyHostToDevice);
		hipMemcpy(dev_indices, scene->meshes[i].indices, scene->meshes[i].numIndices * sizeof(unsigned short), hipMemcpyHostToDevice);

		dev_meshes[i].vertices = dev_vertices;
		dev_meshes[i].indices = dev_indices;
	}

	numTrees = scene->octrees.size();
	hipMallocManaged(&dev_octrees, numTrees * sizeof(OctreeDev));
	hipMemcpy(dev_octrees, scene->octrees.data(), numTrees * sizeof(OctreeDev), hipMemcpyHostToDevice);
	
	for (int i = 0; i < numTrees; i++) {
		OctreeDev& octree = dev_octrees[i];
		octree.root = scene->octrees[i].root;
		octree.materialid = scene->octrees[i].materialid;
		octree.numNodes = scene->octrees[i].nodes.size();
		octree.transform = scene->octrees[i].transform;
		octree.inverseTransform = scene->octrees[i].inverseTransform;
		octree.invTranspose = scene->octrees[i].invTranspose;

		OctreeNode* dev_nodes;
		Triangle* dev_triangles;
		Geom* dev_bounds;
		int* dev_dataStarts;

		hipMalloc(&dev_nodes, octree.numNodes * sizeof(OctreeNode));
		hipMalloc(&dev_triangles, scene->octrees[i].triangles.size() * sizeof(Triangle));
		hipMalloc(&dev_bounds, scene->octrees[i].boundingBoxes.size() * sizeof(Geom));
		hipMalloc(&dev_dataStarts, scene->octrees[i].dataStarts.size() * sizeof(int));

		hipMemcpy(dev_nodes, scene->octrees[i].nodes.data(), octree.numNodes * sizeof(OctreeNode), hipMemcpyHostToDevice);
		hipMemcpy(dev_triangles, scene->octrees[i].triangles.data(), scene->octrees[i].triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);
		hipMemcpy(dev_bounds, scene->octrees[i].boundingBoxes.data(), scene->octrees[i].boundingBoxes.size() * sizeof(Geom), hipMemcpyHostToDevice);
		hipMemcpy(dev_dataStarts, scene->octrees[i].dataStarts.data(), scene->octrees[i].dataStarts.size() * sizeof(int), hipMemcpyHostToDevice);

		dev_octrees[i].nodes = dev_nodes;
		dev_octrees[i].triangles = dev_triangles;
		dev_octrees[i].boundingBoxes = dev_bounds;
		dev_octrees[i].dataStarts = dev_dataStarts;
	}

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	checkCUDAError("pathtraceFree");
	hipFree(dev_paths);
	checkCUDAError("pathtraceFree");
	hipFree(dev_geoms);
	checkCUDAError("pathtraceFree");
	hipFree(dev_materials);
	checkCUDAError("pathtraceFree");
	hipFree(dev_intersections);
	checkCUDAError("pathtraceFree");
	hipFree(dev_intersections_gbuffer);
	checkCUDAError("pathtraceFree");
	for (int i = 0; i < meshSize; i++)
	{
		hipFree(dev_meshes[i].vertices);
		hipFree(dev_meshes[i].indices);
	}
	hipFree(dev_meshes);
	for (int i = 0; i < numTrees; i++) {
		hipFree(dev_octrees[i].nodes);
		hipFree(dev_octrees[i].triangles);
		hipFree(dev_octrees[i].boundingBoxes);
		hipFree(dev_octrees[i].dataStarts);
	}
	hipFree(dev_octrees);
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, Mesh* meshes
	, OctreeDev* octrees
	, int geoms_size
	, int meshes_size
	, int num_trees
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		int hit_mesh_index = -1;
		int hit_tree_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}
		// for (int i = 0; i < meshes_size; i++)
		// {
		// 	Mesh& mesh = meshes[i];

		// 	t = boxIntersectionTest(mesh.boundingVolume, pathSegment.ray, tmp_intersect, tmp_normal, outside);
		// 	if (t <= 0.0f)
		// 	{
		// 		continue;
		// 	}
		// 	t = meshIntersectionTest(mesh, pathSegment.ray, tmp_intersect, tmp_normal, outside);
		// 	// if (t > 0.0f && t_min > t)
		// 	// {
		// 	// 	t_min = t;
		// 	// 	hit_mesh_index = i;
		// 	// 	hit_geom_index = -1;
		// 	// 	hit_tree_index = -1;
		// 	// 	intersect_point = tmp_intersect;
		// 	// 	normal = tmp_normal;
		// 	// }
		// }
		for (int i = 0; i < num_trees; i++)
		{
			OctreeDev& octree = octrees[i];
			//t = boxIntersectionTest(octree.boundingBoxes[3], pathSegment.ray, tmp_intersect, tmp_normal, outside);
			t = octreeIntersectionTest(octree, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_mesh_index = -1;
				hit_geom_index = -1;
				hit_tree_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1 && hit_mesh_index == -1 && hit_tree_index == -1)
		{
			intersections[path_index].t = -1.0f;
			return;
		}
		intersections[path_index].t = t_min;
		intersections[path_index].surfaceNormal = normal;
		intersections[path_index].intersectPoint = intersect_point;
		if (hit_mesh_index != -1)
		{
			intersections[path_index].materialId = meshes[hit_mesh_index].materialid;
		}
		if (hit_geom_index != -1)
		{
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
		}
		if (hit_tree_index != -1)
		{
			intersections[path_index].materialId = octrees[hit_tree_index].materialid;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

__global__ void shadeMaterial(
	int iter,
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	Material* materials
) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths) {
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) {
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				scatterRay(pathSegments[idx], intersection.intersectPoint, intersection.surfaceNormal, material, rng);
				pathSegments[idx].remainingBounces--;
			}
		}
		else {
			pathSegments[idx].color = BACKGROUND_COLOR;
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths, int offset)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x + offset;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

struct ExtractMaterialId
{
    __host__ __device__
    int operator()(const ShadeableIntersection& intersection) const {
        return intersection.materialId;
    }
};


struct is_valid {
	__host__ __device__
		bool operator()(const PathSegment& path) {
		return path.remainingBounces > 0;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {

	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipStream_t streams[nStreams];
	for (int i = 0; i < nStreams; i++)
	{
		hipStreamCreate(&streams[i]);
	}

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / (blockSize2d.y));

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d, 0>> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	// generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	// checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	bool iterationComplete = false;
	hasGBuffer = !(iter == 1) && useGBuffer;

	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		// tracing
		if (!hasGBuffer || depth > 0) {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
						depth
						, num_paths
						, dev_paths
						, dev_geoms
						, dev_meshes
						, dev_octrees
						, hst_scene->geoms.size()
						, hst_scene->meshes.size()
						, hst_scene->octrees.size()
						, dev_intersections
						);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();

			if (!hasGBuffer && useGBuffer) {
				hipMemcpy(dev_intersections_gbuffer, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
				
				hasGBuffer = true;
				checkCUDAError("hipMemcpy");
			}			
		}
		else {
			hipMemcpy(dev_intersections, dev_intersections_gbuffer, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}	

		checkCUDAError("trace one bounce");
		//hipDeviceSynchronize();
		depth++;

		if (sortByMaterial)
		{
			std::cout << "sortByMaterial" << std::endl;
			thrust::device_ptr<PathSegment> dev_paths_ptr(dev_paths);
			thrust::device_ptr<ShadeableIntersection> dev_intersections_ptr(dev_intersections);

			thrust::device_vector<int> materialIds(num_paths);
			thrust::transform(dev_intersections_ptr, dev_intersections_ptr + num_paths, materialIds.begin(), ExtractMaterialId());

			thrust::sort_by_key(thrust::device, materialIds.begin(), materialIds.end(),
								thrust::make_zip_iterator(thrust::make_tuple(dev_paths_ptr, dev_intersections_ptr)));

			checkCUDAError("thrust::sort_by_key");
		}

		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d, blockSize1d * sizeof(ShadeableIntersection) >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths, 
			dev_materials
		    );
		checkCUDAError("shadeMaterial");

		dev_path_end = thrust::partition(thrust::device, dev_paths, dev_path_end, is_valid());
		num_paths = dev_path_end - dev_paths;

		if (num_paths <= 0 || depth >= traceDepth) {
			iterationComplete = true;
		}

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// hipDeviceSynchronize();
	// int streamSize = pixelcount / nStreams;
	// dim3 numBlocksPixels = (streamSize + blockSize1d - 1) / blockSize1d;
	// for (int i = 0; i < nStreams; i++)
	// {
	// 	int offset = i * streamSize;
	// 	finalGather << <numBlocksPixels, blockSize1d, 0, streams[i] >> > (num_paths, dev_image, dev_paths, offset);
	// }
	// // Assemble this iteration and apply it to the image
	// hipDeviceSynchronize();
	// checkCUDAError("finalGather");

	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths,0);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	for (int i = 0; i < nStreams; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	checkCUDAError("pathtrace");
}
