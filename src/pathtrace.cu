#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/sort.h>
#include <thrust/partition.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include<thrust/scan.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define RANDVEC3 glm::vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))


#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

__device__ inline bool isNAN(const glm::vec3& v)
{
	return (v.x != v.x) || (v.y != v.y) || (v.z != v.z);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

class CUDATimer
{
public:
	CUDATimer(const std::string& inName)
	{
		name = inName;

		hipEventCreate(&startEvent);
		hipEventCreate(&stopEvent);
	}

	~CUDATimer()
	{
		hipEventDestroy(startEvent);
		hipEventDestroy(stopEvent);
	}

	void start()
	{
		hipEventRecord(startEvent);
	}

	void stop()
	{
		hipEventRecord(stopEvent);

		hipEventSynchronize(stopEvent);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

		printf("%sIt takes: %f ms\n", name.c_str(), milliseconds);
	}

private:
	hipEvent_t startEvent;
	hipEvent_t stopEvent;
	std::string name;
};

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static int* dev_materialSortBuffer = nullptr;
static int* dev_materialSortBuffer2 = nullptr;
static ShadeableIntersection* dev_intersections = NULL;

int samplesPerPixel = 1;

bool sortMaterial = false;

int* dev_perm_x = nullptr;
int* dev_perm_y = nullptr;
int* dev_perm_z = nullptr;

perlin** dev_perlinNoise = nullptr;

bool perlinInitialized = false;

// TODO: static variables for device memory, any extra info you need, etc
// ...
static ShadeableIntersection* dev_cache = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

__global__ void initPerlin(perlin** perlinNoise, thrust::default_random_engine rng, int* dev_perm_x, int* dev_perm_y, int* dev_perm_z)
{
	*perlinNoise = new perlin(rng, dev_perm_x, dev_perm_y, dev_perm_z);
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	samplesPerPixel = guiData->SamplePerPixel;

	//hst_scene->state.traceDepth = guiData->Depth;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, samplesPerPixel * pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_materialSortBuffer, samplesPerPixel * pixelcount * sizeof(int));
	hipMalloc(&dev_materialSortBuffer2, samplesPerPixel * pixelcount * sizeof(int));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, samplesPerPixel * pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, samplesPerPixel * pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_cache, samplesPerPixel * pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_cache, 0, samplesPerPixel * pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_perm_x, 256 * sizeof(int));
	hipMemset(dev_perm_x, 0, 256 * sizeof(int));

	hipMalloc(&dev_perm_y, 256 * sizeof(int));
	hipMemset(dev_perm_y, 0, 256 * sizeof(int));

	hipMalloc(&dev_perm_z, 256 * sizeof(int));
	hipMemset(dev_perm_z, 0, 256 * sizeof(int));

	hipMalloc(&dev_perlinNoise, sizeof(perlin*));

	thrust::default_random_engine rng = makeSeededRandomEngine(0, 1, 2);

	initPerlin<<<1, 1>>>(dev_perlinNoise, rng, dev_perm_x, dev_perm_y, dev_perm_z);

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_materialSortBuffer);
	hipFree(dev_materialSortBuffer2);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_cache);
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int samplesPerPixel)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		//PathSegment& segment = pathSegments[index];

		//segment.ray.origin = cam.position;
		//segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		//segment.ray.direction = glm::normalize(cam.view
		//	- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
		//	- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f));

		//segment.pixelIndex = index;
		//segment.remainingBounces = traceDepth;

		// TODO: implement antialiasing by jittering the ray
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
		thrust::uniform_real_distribution<float> u01(0, 1);

		for (int i = 0; i < samplesPerPixel; i++)
		{
			int jitterIndex = cam.resolution.x * cam.resolution.y * i + index;

			PathSegment& segment = pathSegments[jitterIndex];

			segment.ray.origin = cam.position;
			segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

			glm::vec2 jitter = glm::vec2(0.5f * (u01(rng) * 2.0f - 1.0f), 0.5f * (u01(rng) * 2.0f - 1.0f));

			if (i == 0)
			{
				jitter = glm::vec2(0.0f);
			}

			pathSegments[jitterIndex].ray.direction = glm::normalize(cam.view
				- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + jitter[0])
				- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + jitter[1]));

			segment.pixelIndex = index;
			segment.remainingBounces = traceDepth;
		}
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int maxDepth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
	, glm::vec3* image
	, hipTextureObject_t skyboxTex
	, int imageWidth
	, int imageHeight
	, int samplesPerPixel
	, int* materialKeys
)
{
	int pathIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (pathIndex >= num_paths)
	{
		return;
	}

	for (int i = 0; i < samplesPerPixel; i++)
	{
		int jitterIndex = imageWidth * imageHeight * i + pathIndex;

		PathSegment& pathSegment = pathSegments[jitterIndex];

		intersections[jitterIndex].materialId = -1;

		float t;
		glm::vec3 intersectPoint;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;
		float u;
		float v;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, u, v);
			}
			else if (geom.type == MESH)
			{
				t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == PROCEDURAL)
			{
				t = proceduralIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersectPoint = tmp_intersect;
				normal = tmp_normal;
			}
		}

		int materialId = -1;
		if (hit_geom_index == -1)
		{
			intersections[jitterIndex].t = -1.0f;

			if (pathSegment.remainingBounces == maxDepth)
			{
				pathSegment.needSkyboxColor = true;
			}

			pathSegment.remainingBounces = 0;
		}
		else
		{
			//The ray hits something
			intersections[jitterIndex].t = t_min;
			intersections[jitterIndex].materialId = geoms[hit_geom_index].materialid;
			intersections[jitterIndex].surfaceNormal = normal;
			intersections[jitterIndex].frontFace = outside;
			intersections[jitterIndex].point = intersectPoint;
			intersections[jitterIndex].u = u;
			intersections[jitterIndex].v = v;
			materialId = intersections[jitterIndex].materialId;
		}
		materialKeys[jitterIndex] = materialId;
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int numPaths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
	, int imageWidth
	, int imageHeight
	, int samplesPerPixel
	, perlin** perlinNoise
	, int* dev_perm_x
	, int* dev_perm_y
	, int* dev_perm_z
	, hipTextureObject_t skyBoxTexture
)
{
	int pathIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (pathIndex >= numPaths)
	{
		return;
	}

	// Set up the RNG
	// LOOK: this is how you use thrust's RNG! Please look at
	// makeSeededRandomEngine as well.
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, pathIndex, 0);
	thrust::uniform_real_distribution<float> u01(0, 1);

	// getPointOnRay(pathSegments[idx].ray, intersection.t)

	glm::vec3 color = glm::vec3(0.0f);

	for (int i = 0; i < samplesPerPixel; i++)
	{
		int jitterIndex = imageWidth * imageHeight * i + pathIndex;

		if (pathSegments[jitterIndex].remainingBounces == 0)
		{
			//glm::vec3 unitDirection = glm::normalize(pathSegments[jitterIndex].ray.direction);
			//float t = 0.5f * (unitDirection.y + 1.0f);
			//glm::vec3 backgroundColor = (1.0f - t) * glm::vec3(1.0, 1.0, 1.0) + t * glm::vec3(0.5, 0.7, 1.0);
			//pathSegments[jitterIndex].color *= backgroundColor;
			glm::vec2 uv = sampleHDRMap(glm::normalize(pathSegments[jitterIndex].ray.direction));
			float4 skyColorRGBA = tex2D<float4>(skyBoxTexture, uv.x, uv.y);
			glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
			//pathSegments[jitterIndex].color *= skyColor;

			color += pathSegments[jitterIndex].color;

			continue;
		}

		ShadeableIntersection intersection = shadeableIntersections[jitterIndex];

		Material material;

		if (intersection.materialId >= 0)
		{
			material = materials[intersection.materialId];
		}

		scatterRay(pathSegments[jitterIndex], intersection, intersection.point, intersection.surfaceNormal, intersection.frontFace, material, rng);

		if (material.pattern == Pattern::PerlinNoise)
		{
			glm::vec3 perlinNoiseColor = glm::vec3(1.0f, 1.0f, 1.0f) * 0.5f * 
										  (1.0f + glm::sin(1.0f * intersection.point.z + 10.0f * 
										  (*perlinNoise)->turb(intersection.point, 7, dev_perm_x, dev_perm_y, dev_perm_z)));
			pathSegments[jitterIndex].color *= perlinNoiseColor;
		}

		color += pathSegments[jitterIndex].color;
	}

	color /= samplesPerPixel;

	pathSegments[pathIndex].color = color;
	//// If the material indicates that the object was a light, "light" the ray
	//if (material.emittance > 0.0f) {
	//	pathSegments[idx].color *= (materialColor * material.emittance);
	//}
	//// Otherwise, do some pseudo-lighting computation. This is actually more
	//// like what you would expect from shading in a rasterizer like OpenGL.
	//// TODO: replace this! you should be able to start with basically a one-liner
	//else {
	//	float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
	//	pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
		//pathSegments[idx].color *= u01(rng); // apply some noise because why not
	//}
	//// If there was no intersection, color the ray black.
	//// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
	//// used for opacity, in which case they can indicate "no opacity".
	//// This can be useful for post-processing and image compositing.
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int numPaths, glm::vec3* image, PathSegment* iterationPaths, hipTextureObject_t skyboxTexture)
{
	int pathIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (pathIndex < numPaths)
	{
		PathSegment iterationPath = iterationPaths[pathIndex];
		if (!isNAN(iterationPath.color))
		{
			glm::vec2 uv = sampleHDRMap(glm::normalize(iterationPath.ray.direction));
			float4 skyColorRGBA = tex2D<float4>(skyboxTexture, uv.x, uv.y);
			glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);

			if (iterationPath.needSkyboxColor)
			{
				image[iterationPath.pixelIndex] += iterationPath.color * skyColor * 3.0f;
			}
			else
			{
				image[iterationPath.pixelIndex] += iterationPath.color;
			}
		}
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths, samplesPerPixel);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	int current_num_paths = num_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;

	while (!iterationComplete) {
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

		if (depth == 0)
		{
			if (iter == 1) {
				// tracing
				computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth
					, traceDepth
					, num_paths
					, dev_paths
					, dev_geoms
					, hst_scene->geoms.size()
					, dev_intersections
					, dev_image
					, hst_scene->skyboxTextureObject
					, cam.resolution.x
					, cam.resolution.y
					, samplesPerPixel
					, dev_materialSortBuffer
					);
				checkCUDAError("trace one bounce");
				hipDeviceSynchronize();
				hipMemcpy(dev_cache, dev_intersections, samplesPerPixel * pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
			else {
				hipMemcpy(dev_intersections, dev_cache, samplesPerPixel * pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}

		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, traceDepth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				, dev_image
				, hst_scene->skyboxTextureObject
				, cam.resolution.x
				, cam.resolution.y
				, samplesPerPixel
				, dev_materialSortBuffer
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}
		depth++;

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			cam.resolution.x,
			cam.resolution.y,
			samplesPerPixel,
			dev_perlinNoise,
			dev_perm_x,
			dev_perm_y,
			dev_perm_z,
			hst_scene->skyboxTextureObject
			);

		//CUDATimer timer("shadeFakeMaterial");

		//timer.start();

		if (sortMaterial)
		{
			//thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + current_num_paths, dev_paths, SortIntersection());
			hipMemcpy(dev_materialSortBuffer2, dev_materialSortBuffer, sizeof(int) * num_paths, hipMemcpyDeviceToDevice);
			thrust::sort_by_key(thrust::device, dev_materialSortBuffer, dev_materialSortBuffer + num_paths, dev_intersections);
			thrust::sort_by_key(thrust::device, dev_materialSortBuffer2, dev_materialSortBuffer2 + num_paths, dev_paths);

			//timer.stop();

			//dev_path_end = thrust::stable_partition(thrust::device, dev_paths, dev_path_end, returnRemainBounce());
			dev_path_end = thrust::partition(thrust::device, dev_paths, dev_path_end, returnRemainBounce());
			current_num_paths = dev_path_end - dev_paths;

			//printf("%d\n", current_num_paths);

			iterationComplete = (depth >= traceDepth || current_num_paths <= 0);
		}
		else
		{
			if (depth == traceDepth)
			{
				iterationComplete = true; // TODO: should be based off stream compaction results.
			}
		}

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;

			samplesPerPixel = guiData->SamplePerPixel;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths, hst_scene->skyboxTextureObject);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
