#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define SORT_BY_MATERIAL 0
#define CACHE_FIRST_BOUNCE 1
#define ANTI_ALIASING 1
#define DEPTH_OF_FIELD 1
#define OIDN 1

#if ANTI_ALIASING
#undef CACHE_FIRST_BOUNCE
#define CACHE_FIRST_BOUNCE 0
#endif // ANTI_ALIASING
#if DEPTH_OF_FIELD
#undef CACHE_FIRST_BOUNCE
#define CACHE_FIRST_BOUNCE 0
#endif // DEPTH_OF_FIELD
#if OIDN
#define EMA_ALPHA 0.2f
#define DENOISE_INTERVAL 20
#endif // OIDN

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);

		glm::ivec3 color = glm::clamp(
			image[index] * 255.0f / (float)iter,
			glm::vec3(0.0f), glm::vec3(255.0f));

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
		pbo[index].w = 0;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
#if CACHE_FIRST_BOUNCE
static ShadeableIntersection* dev_first_bounce_intersections = NULL;
#endif // CACHE_FIRST_BOUNCE

static glm::vec3* dev_vertices = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec2* dev_texcoords = NULL;
static Mesh* dev_meshes = NULL;

#if OIDN
#include "OpenImageDenoise/oidn.hpp"

static glm::vec3* dev_denoised = NULL;
static glm::vec3* dev_albedo = NULL;
static glm::vec3* dev_normal = NULL;

void denoise()
{
	int width = hst_scene->state.camera.resolution.x,
		height = hst_scene->state.camera.resolution.y;

	// Create an Intel Open Image Denoise device
    oidn::DeviceRef device = oidn::newDevice();
    device.commit();

    // Create buffers for input/output images accessible by both host (CPU) and device (CPU/GPU)
	// oidn::BufferRef colorBuf = device.newBuffer(width * height * sizeof(glm::vec3));
	// oidn::BufferRef albedoBuf = device.newBuffer(width * height * sizeof(glm::vec3));
	// oidn::BufferRef normalBuf = device.newBuffer(width * height * sizeof(glm::vec3));

    // Create a filter for denoising a beauty (color) image using prefiltered auxiliary images too
    oidn::FilterRef filter = device.newFilter("RT"); // generic ray tracing filter
    filter.setImage("color", dev_image, oidn::Format::Float3, width, height); // beauty
	filter.setImage("albedo", dev_albedo, oidn::Format::Float3, width, height); // auxiliary
	filter.setImage("normal", dev_normal, oidn::Format::Float3, width, height); // auxiliary
    filter.setImage("output", dev_denoised, oidn::Format::Float3, width, height); // denoised beauty
    filter.set("hdr", true); // image is HDR
	filter.set("cleanAux", true); // auxiliary images will be prefiltered
    filter.commit();

	// Copy the rendered image to the color buffer
	// hipMemcpy(colorBuf.getData(), dev_image, width * height * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	// hipMemcpy(albedoBuf.getData(), dev_albedo, width * height * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	// hipMemcpy(normalBuf.getData(), dev_normal, width * height * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	// hipDeviceSynchronize();

	// Create a separate filter for denoising an auxiliary albedo image (in-place)
	oidn::FilterRef albedoFilter = device.newFilter("RT"); // same filter type as for beauty
	albedoFilter.setImage("albedo", dev_albedo, oidn::Format::Float3, width, height);
	albedoFilter.setImage("output", dev_albedo, oidn::Format::Float3, width, height);
	albedoFilter.commit();

	// Create a separate filter for denoising an auxiliary normal image (in-place)
	oidn::FilterRef normalFilter = device.newFilter("RT"); // same filter type as for beauty
	normalFilter.setImage("normal", dev_normal, oidn::Format::Float3, width, height);
	normalFilter.setImage("output", dev_normal, oidn::Format::Float3, width, height);
	normalFilter.commit();

	// Prefilter the auxiliary images
	albedoFilter.execute();
	normalFilter.execute();

	// Filter the beauty image
	filter.execute();

	// Check for errors
	const char* errorMessage;
	if (device.getError(errorMessage) != oidn::Error::None)
		std::cout << "Error: " << errorMessage << std::endl;
}

__global__
void copyFirstTraceToAlbedoAndNormal(
	PathSegment *pathSegments, int num_paths,
	ShadeableIntersection *shadeableIntersections,
	glm::vec3 *albedo, glm::vec3 *normal)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths) {
		PathSegment pathSegment = pathSegments[idx];
		ShadeableIntersection intersection = shadeableIntersections[idx];

		albedo[pathSegment.pixelIndex] = pathSegment.color;
		normal[pathSegment.pixelIndex] = intersection.surfaceNormal;
	}
}

__global__
void emaMergeDenoisedAndImage(int pixelcount, glm::vec3 *image, glm::vec3 *denoised)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < pixelcount) {
		// exponential moving average
		image[idx] = image[idx] * (1 - EMA_ALPHA) + denoised[idx] * EMA_ALPHA;
	}
}

#endif // OIDN

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
#if CACHE_FIRST_BOUNCE
	hipMalloc(&dev_first_bounce_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_bounce_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
#endif // CACHE_FIRST_BOUNCE

	hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(glm::vec3));
	hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

	hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
	hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

	hipMalloc(&dev_texcoords, scene->texcoords.size() * sizeof(glm::vec2));
	hipMemcpy(dev_texcoords, scene->texcoords.data(), scene->texcoords.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);

	hipMalloc(&dev_meshes, scene->meshes.size() * sizeof(Mesh));
	hipMemcpy(dev_meshes, scene->meshes.data(), scene->meshes.size() * sizeof(Mesh), hipMemcpyHostToDevice);

#if OIDN
	hipMalloc(&dev_denoised, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_denoised, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_albedo, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_normal, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_normal, 0, pixelcount * sizeof(glm::vec3));
#endif // OIDN

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created

#if CACHE_FIRST_BOUNCE
	hipFree(dev_first_bounce_intersections);
#endif // CACHE_FIRST_BOUNCE

	hipFree(dev_vertices);
	hipFree(dev_normals);
	hipFree(dev_texcoords);
	hipFree(dev_meshes);

#if OIDN
	hipFree(dev_denoised);
	hipFree(dev_albedo);
	hipFree(dev_normal);
#endif // OIDN

	checkCUDAError("pathtraceFree");
}

#if DEPTH_OF_FIELD
__host__ __device__
glm::vec2 ConcentricSampleDisk(const glm::vec2& u)
{
	glm::vec2 uOffset = 2.0f * u - glm::vec2(1.0f, 1.0f);

	if (uOffset.x == 0.0f && uOffset.y == 0.0f)
	{
		return glm::vec2(0.0f, 0.0f);
	}

	float theta, r;
	if (glm::abs(uOffset.x) > glm::abs(uOffset.y))
	{
		r = uOffset.x;
		theta = PI_OVER_FOUR * (uOffset.y / uOffset.x);
	}
	else
	{
		r = uOffset.y;
		theta = PI_OVER_TWO - PI_OVER_FOUR * (uOffset.x / uOffset.y);
	}
	return r * glm::vec2(glm::cos(theta), glm::sin(theta));
}
#endif // DEPTH_OF_FIELD

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__
void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	// locate (x, y) in pixel space
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// make sure (x, y) is not out of bounds
	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);
#if ANTI_ALIASING
		// implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
		);
#else
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#endif // ANTI_ALIASING
#if DEPTH_OF_FIELD
		if (cam.lensRadius > 0)
		{
			glm::vec2 pLens = cam.lensRadius * ConcentricSampleDisk(glm::vec2(u01(rng), u01(rng)));
			float ft = cam.focalDistance / glm::dot(cam.view, segment.ray.direction);
			glm::vec3 pFocus = segment.ray.origin + segment.ray.direction * ft;
			segment.ray.origin += cam.right * pLens.x + cam.up * pLens.y;
			segment.ray.direction = glm::normalize(pFocus - segment.ray.origin);
		}
#endif // DEPTH_OF_FIELD

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__
void computeIntersections(
	int depth,
	PathSegment* pathSegments,
	int num_paths,
	Geom* geoms,
	int geoms_size,
	Mesh* meshes,
	glm::vec3* vertices,
	glm::vec3* normals,
	glm::vec2* texcoords,
	ShadeableIntersection* intersections)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_material_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		int tmp_material_index;
		glm::vec3 tmp_normal;
		glm::vec2 tmp_texcoord;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				tmp_material_index = geom.materialid;
				tmp_normal = outside ? tmp_normal : -tmp_normal;
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				tmp_material_index = geom.materialid;
				tmp_normal = outside ? tmp_normal : -tmp_normal;
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?
			else if (geom.type == MESH)
			{
				t = meshIntersectionTest(geom, meshes, vertices, normals, texcoords, pathSegment.ray,
										 tmp_intersect, tmp_material_index, tmp_normal, tmp_texcoord);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_material_index = tmp_material_index;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_material_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = hit_material_index;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__
void shadePhysicallyBasedMaterial(
	int iter,
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	Material* materials)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths) {
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) {
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);

			Material material = materials[intersection.materialId];

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (material.albedo * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				scatterRay(pathSegments[idx], getPointOnRay(pathSegments[idx].ray, intersection.t),
					intersection.surfaceNormal, material, rng);
				if (pathSegments[idx].remainingBounces == 0) pathSegments[idx].color = glm::vec3(0.0f);
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		} else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

// Predicate for stream compaction
struct gtZero
{
	__host__ __device__
	bool operator()(const PathSegment &x)
	{
		return x.remainingBounces > 0;
	}
};

struct compareMaterialId
{
	__host__ __device__
	bool operator()(const ShadeableIntersection &x, const ShadeableIntersection &y)
	{
		return x.materialId < y.materialId;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if CACHE_FIRST_BOUNCE
		if (depth == 0 && iter > 1)
		{
			hipMemcpy(dev_intersections, dev_first_bounce_intersections,
					   pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			checkCUDAError("copy first bounce");
		}
		else
		{
			// clean shading chunks
			hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
			// tracing
			computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
				depth,
				dev_paths, num_paths,
				dev_geoms, hst_scene->geoms.size(),
				dev_meshes, dev_vertices, dev_normals, dev_texcoords,
				dev_intersections);
			// cache first bounce
			if (depth == 0 && iter == 1)
			{
				hipMemcpy(dev_first_bounce_intersections, dev_intersections,
						   pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
			checkCUDAError("trace one bounce");
		}
		hipDeviceSynchronize();
#else
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		// tracing
		computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
			depth,
			dev_paths, num_paths,
			dev_geoms, hst_scene->geoms.size(),
			dev_meshes, dev_vertices, dev_normals, dev_texcoords,
			dev_intersections);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
#endif // CACHE_FIRST_BOUNCE
		depth++;

#if SORT_BY_MATERIAL
		// sort paths by material
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, compareMaterialId());
#endif // SORT_BY_MATERIAL

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.
		shadePhysicallyBasedMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
			iter, num_paths, dev_intersections, dev_paths, dev_materials);

#if OIDN
		// copy image to albedo and normal
		if (depth == 1)
			copyFirstTraceToAlbedoAndNormal<<<numblocksPathSegmentTracing, blockSize1d>>>(
				dev_paths, num_paths, dev_intersections, dev_albedo, dev_normal);
#endif // OIDN

		// iterationComplete = true; // TODO: should be based off stream compaction results.
		dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, gtZero());
		num_paths = dev_path_end - dev_paths;
		iterationComplete = num_paths == 0;

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

#if OIDN
	if (iter % DENOISE_INTERVAL == 0)
	{
		denoise();
		emaMergeDenoisedAndImage<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_denoised);
	}
#endif // OIDN

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
