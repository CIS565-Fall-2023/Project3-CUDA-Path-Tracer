#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <numeric>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static PathSegment* dev_tempPaths = NULL;
static PathSegment* dev_pathsBuffer = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static ShadeableIntersection* dev_tempIntersections = NULL;
static ShadeableIntersection* dev_intersectionsBuffer = NULL;
static int* dev_bools = NULL;
static int* dev_nbools = NULL;
static int* dev_scanBools = NULL;
static int* dev_scanNBools = NULL;
static Triangle* dev_tris = NULL;
static BoundingBox* dev_bvh = NULL;
static TriangleArray* dev_triArr = NULL;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void printArr(int n, int* odata, int* dev_odata) {
	hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy dev_odata -> odata for printArr failed!");
	for (int i = 0; i <= n / 10; i++) {
		for (int j = 0; j < 10 && j < n - 10 * i; j++) {
			std::cout << odata[i * 10 + j] << "  ";
		}std::cout << std::endl;
	}std::cout << std::endl << std::endl;
}

void printArr(int begin, int n, int* dev_odata) {
	int o[10];
	for (int i = 0; i <= n / 10; i++) {
		hipMemcpy(o, dev_odata + begin + 10 * i, sizeof(int) * 10, hipMemcpyDeviceToHost);
		if (o[0]+o[1]+o[2]+o[3]+o[4]+o[5]+o[6]+o[7]+o[8]+o[9] == -10) {
			continue;
		}
		checkCUDAError("hipMemcpy dev_odata -> odata for printArr failed!");
		for (int j = 0; j < 10 && j < n - 10 * i; j++) {
			std::cout << o[j] << "  ";
		}std::cout << std::endl;
	}std::cout << std::endl << std::endl;
}

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_tempPaths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_tris, scene->tris.size() * sizeof(Triangle));
	hipMemcpy(dev_tris, scene->tris.data(), scene->tris.size() * sizeof(Triangle), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	hipMalloc(&dev_tempIntersections, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_bools, pixelcount * sizeof(int));
	hipMalloc(&dev_nbools, pixelcount * sizeof(int));
	hipMalloc(&dev_scanBools, pixelcount * sizeof(int));
	hipMalloc(&dev_scanNBools, pixelcount * sizeof(int));

#ifdef CACHE_FIRST_BOUNCE
	hipMalloc(&dev_intersectionsBuffer, pixelcount * sizeof(ShadeableIntersection));
	hipMalloc(&dev_pathsBuffer, pixelcount * sizeof(PathSegment));
#endif

#ifdef USING_BVH
	hipMalloc(&dev_bvh, scene->bvh.size() * sizeof(BoundingBox));
	hipMemcpy(dev_bvh, scene->bvh.data(), scene->bvh.size() * sizeof(BoundingBox), hipMemcpyHostToDevice);
	hipMalloc(&dev_triArr, scene->triArr.size() * sizeof(TriangleArray));
	hipMemcpy(dev_triArr, scene->triArr.data(), scene->triArr.size() * sizeof(TriangleArray), hipMemcpyHostToDevice);
#endif

	checkCUDAError("pathtraceInit");	
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_tris);
	hipFree(dev_materials);
	hipFree(dev_intersections);

	hipFree(dev_bools);
	hipFree(dev_nbools);
	hipFree(dev_scanBools);
	hipFree(dev_scanNBools);
	hipFree(dev_tempPaths);
	hipFree(dev_tempIntersections);
#ifdef CACHE_FIRST_BOUNCE
	hipFree(dev_intersectionsBuffer);
	hipFree(dev_pathsBuffer);
#endif
#ifdef USING_BVH
	hipFree(dev_bvh);
	hipFree(dev_triArr);
#endif
	checkCUDAError("pathtraceFree");
}



/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersectionsNaive(
	int depth
	, int num_paths
	, const PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, Triangle* tris
	, int tris_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_index = -1;
		bool hit_geom = true;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		for (int i = 0; i < tris_size; i++) {
			Triangle& tri = tris[i];
			t = triangleIntersectionTest(tri, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_index = i;
				hit_geom = false;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = hit_geom ? geoms[hit_index].materialid : tris[hit_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}


__device__ __host__ void searchTriArrIntersect(
	Ray& ray, float& t_min, int& hit_index, TriangleArray& triIndices, Triangle* tris,
	glm::vec3& intersectionPoint, glm::vec3& normal, bool& outside
) {
	#pragma unroll
	for (int j = 0; j < BBOX_TRI_NUM; j++) {
		int ti = triIndices.triIds[j];
		if (ti < 0) { return; }
		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;
		float t = triangleIntersectionTest(tris[ti], ray, tmp_intersect, tmp_normal, outside);
		if (t > 0.0f && t_min > t)
		{
			t_min = t;
			hit_index = ti;
			intersectionPoint = tmp_intersect;
			normal = tmp_normal;
		}
	}
}

__device__ __host__ float bvhSearch(
	Ray& ray, int& hit_index
	, Triangle* tris, int tris_size
	, BoundingBox* bvh, int bvh_size
	, TriangleArray* tri_arr, int tri_arr_size
	, glm::vec3& intersectionPoint, glm::vec3& normal, bool& outside
) {

	float t_min = 1e5;

	// Test BVH
	//for (int i = 0; i < bvh_size; i++) {
	//	BoundingBox& bbox = bvh[i];
	//	// if (boundingboxIntersectionTest(ray, bbox.min, bbox.max))
	//	
	//	// reach leaf node of bvh
	//	if (bbox.triArrId >= 0) {
	//		searchTriArrIntersect(ray, t_min, hit_index, tri_arr[bbox.triArrId], tris, intersectionPoint, normal, outside);
	//	}
	//}

	

	
	int arr[BVH_GPU_STACK_SIZE];
	int sign = 0;
	arr[0] = 0;

	while (sign >= 0) {
		BoundingBox& bbox = bvh[arr[sign]];
		sign--;
		if (boundingboxIntersectionTest(ray, bbox.min, bbox.max))
		{
			// reach leaf node of bvh
			if (bbox.triArrId >= 0) {
				searchTriArrIntersect(ray, t_min, hit_index, tri_arr[bbox.triArrId], tris, intersectionPoint, normal, outside);
			}
			// keep searching
			else {
				if (sign + 2 < BVH_GPU_STACK_SIZE) {
					sign++;
					arr[sign] = bbox.leftId;
					sign++;
					arr[sign] = bbox.rightId;
				}
			}
		}
	}
	
	
	return hit_index < 0 ? -1 : t_min;

}


__global__ void computeIntersectionsBVH(
	int depth
	, int num_paths
	, const PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, Triangle* tris
	, int tris_size
	, BoundingBox* bvh
	, int bvh_size
	, TriangleArray* tri_arr
	, int tri_arr_size
	, ShadeableIntersection* intersections
) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_index = -1;
		bool hit_geom = true;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		int hit_tri_index = -1;

		t = bvhSearch(pathSegment.ray, hit_tri_index, tris, tris_size, bvh, bvh_size, tri_arr, tri_arr_size, tmp_intersect, tmp_normal, outside);
		if (t > 0.0f && t_min > t)
		{
			t_min = t;
			hit_index = hit_tri_index;
			hit_geom = false;
			intersect_point = tmp_intersect;
			normal = tmp_normal;
		}


		if (hit_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = hit_geom ? geoms[hit_index].materialid : tris[hit_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}



// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
		}
		else {
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

__global__ void shadeMaterial(
	int iter
	, int num_paths
	, int depth
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths && pathSegments[idx].remainingBounces > 0)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
			// Set up the RNG
			// LOOK: this is how you use thrust's RNG! Please look at
			// makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(depth, idx, iter);
			// thrust::uniform_real_distribution<float> u01(0, 1); // u01(rng) to get random (0, 1)

			Material material = materials[intersection.materialId];

			glm::vec3 intersect = pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * intersection.t;
			scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = -1;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color * (float)abs(iterationPath.remainingBounces);
		/*if (iterationPath.remainingBounces == 0) {
			image[iterationPath.pixelIndex] += glm::vec3(1.0f, 0.0f, 1.0f);
		}
		else {
			image[iterationPath.pixelIndex] += iterationPath.color * (float)abs(iterationPath.remainingBounces);
		}*/
		
	}
}

__global__ void markPathSegment(int nPaths, int* bools, int* nbools, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		bool b = iterationPaths[index].remainingBounces > 0;
		bools[index] = b;
		nbools[index] = !b;
	}
}

// for path termination
__global__ void pathSegmentScatter(int n, int scanSum, PathSegment* odata, const PathSegment* idata, const int* bools, const int* indicesPos, const int* indicesNeg) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		if (bools[index] > 0) {
			odata[indicesPos[index]] = idata[index];
		}
		else {
			odata[indicesNeg[index] + scanSum] = idata[index];
		}
	}
}

// for material sort
__global__ void kernMapMatBitToBoolean(int n, int i, int* bools, int* ebools, const ShadeableIntersection* idata) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		bools[index] = ((idata[index].materialId >> i) & 1);
		ebools[index] = !((idata[index].materialId >> i) & 1);
	}
}

// for path termination
__global__ void pathSegmentAndIntersectionScatter(
	int n, int negCount,
	PathSegment* opaths, const PathSegment* ipaths,
	ShadeableIntersection* ointers, const ShadeableIntersection* iinters,
	const int* bools, const int* indicesPos, const int* indicesNeg) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		if (bools[index] > 0) {
			opaths[indicesPos[index] + negCount] = ipaths[index];
			ointers[indicesPos[index] + negCount] = iinters[index];
		}
		else {
			opaths[indicesNeg[index]] = ipaths[index];
			ointers[indicesNeg[index]] = iinters[index];
		}
	}
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
 ///////////////////////////////////////////////////////////////////////////

 // Recap:
 // * Initialize array of path rays (using rays that come out of the camera)
 //   * You can pass the Camera object to that kernel.
 //   * Each path ray must carry at minimum a (ray, color) pair,
 //   * where color starts as the multiplicative identity, white = (1, 1, 1).
 //   * This has already been done for you.
 // * For each depth:
 //   * Compute an intersection in the scene for each path ray.
 //     A very naive version of this has been implemented for you, but feel
 //     free to add more primitives and/or a better algorithm.
 //     Currently, intersection distance is recorded as a parametric distance,
 //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
 //     * Color is attenuated (multiplied) by reflections off of any object
 //   * TODO: Stream compact away all of the terminated paths.
 //     You may use either your implementation or `thrust::remove_if` or its
 //     cousins.
 //     * Note that you can't really use a 2D kernel launch any more - switch
 //       to 1D.
 //   * TODO: Shade the rays that intersected something or didn't bottom out.
 //     That is, color the ray by performing a color computation according
 //     to the shader, then generate a new ray to continue the ray path.
 //     We recommend just updating the ray's PathSegment in place.
 //     Note that this step may come before or after stream compaction,
 //     since some shaders you write may also cause a path to terminate.
 // * Finally, add this iteration's results to the image. This has been done
 //   for you.
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	int depth = 0;
	int num_paths = pixelcount;

#ifdef CACHE_FIRST_BOUNCE
	if (iter == 1) {
		generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
		checkCUDAError("generate camera ray");
		hipMemcpy(dev_pathsBuffer, dev_paths, sizeof(PathSegment) * pixelcount, hipMemcpyDeviceToDevice);
		checkCUDAError("save dev_pathsBuffer");
	}
	else {
		hipMemcpy(dev_paths, dev_pathsBuffer, sizeof(PathSegment) * pixelcount, hipMemcpyDeviceToDevice);
		checkCUDAError("load dev_pathsBuffer");
	}
#else
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");
#endif



	//Ray testRay = Ray();
	//testRay.origin = cam.position;
	//testRay.direction = glm::normalize(glm::vec3(0, 5, -5) - cam.position);
	//int testTriID = -1;
	//glm::vec3 testIntersectionPoint;
	//glm::vec3 testNormal;
	//bool testOutside;
	//bvhSearch(testRay, testTriID, hst_scene->tris.data(), hst_scene->tris.size(), hst_scene->bvh.data(), hst_scene->bvh.size(),
	//	hst_scene->triArr.data(), hst_scene->triArr.size(), testIntersectionPoint, testNormal, testOutside);

	//cout << "Test: triId = " << testTriID << endl;


	
	// PathSegment* dev_path_end = dev_paths + pixelcount;
	// int num_paths = dev_path_end - dev_paths;
	
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	int mat_num = hst_scene->materials.size();

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {
		
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		checkCUDAError("hipMemset dev_intersections");

#ifdef CACHE_FIRST_BOUNCE
		if (iter > 1 && depth == 0) {
			hipMemcpy(dev_intersections, dev_intersectionsBuffer, sizeof(ShadeableIntersection) * pixelcount, hipMemcpyDeviceToDevice);
			checkCUDAError("load dev_intersectionsBuffer");
		}
		else {
#endif
			// tracing
#ifdef USING_BVH
			if (hst_scene->bvh.size() > 0) {
				computeIntersectionsBVH << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth, num_paths, dev_paths,
					dev_geoms, hst_scene->geoms.size(),
					dev_tris, hst_scene->tris.size(),
					dev_bvh, hst_scene->bvh.size(),
					dev_triArr, hst_scene->triArr.size(),
					dev_intersections);
				checkCUDAError("tcomputeIntersectionsBVH");
			} else {
#endif
				computeIntersectionsNaive << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth, num_paths, dev_paths,
					dev_geoms, hst_scene->geoms.size(),
					dev_tris, hst_scene->tris.size(),
					dev_intersections);
#ifdef USING_BVH
			}
#endif


			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
#ifdef CACHE_FIRST_BOUNCE
			if (iter == 1 && depth == 0) {
				hipMemcpy(dev_intersectionsBuffer, dev_intersections, sizeof(ShadeableIntersection) * pixelcount, hipMemcpyDeviceToDevice);
				checkCUDAError("save dev_intersectionsBuffer");
			}
		}
#endif

		depth++;

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.


		// Sort All Materials
#ifdef MATERIAL_SORT
		if (mat_num > 1) {
			int log2Ceil = 1;
			int product = 1;
			while (product < mat_num) {
				product *= 2;
				log2Ceil++;
			}
			for (int i = 0; i < log2Ceil; i++) {
				kernMapMatBitToBoolean << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, i, dev_bools, dev_nbools, dev_intersections);
				thrust::device_ptr<int> dv_sort_in(dev_bools);
				thrust::device_ptr<int> dv_sort_out(dev_scanBools);
				thrust::exclusive_scan(dv_sort_in, dv_sort_in + num_paths, dv_sort_out);
				thrust::device_ptr<int> dv_sort_nin(dev_nbools);
				thrust::device_ptr<int> dv_sort_nout(dev_scanNBools);
				thrust::exclusive_scan(dv_sort_nin, dv_sort_nin + num_paths, dv_sort_nout);

				int neg_count = -1;
				hipMemcpy(&neg_count, dev_scanNBools + num_paths - 1, sizeof(int), hipMemcpyDeviceToHost);
				int last_bool = -1;
				hipMemcpy(&last_bool, dev_bools + num_paths - 1, sizeof(int), hipMemcpyDeviceToHost);
				neg_count += (last_bool == 0);

				pathSegmentAndIntersectionScatter << < numblocksPathSegmentTracing, blockSize1d >> > (
					num_paths, neg_count,
					dev_tempPaths, dev_paths,
					dev_tempIntersections, dev_intersections,
					dev_bools, dev_scanBools, dev_scanNBools);

				PathSegment* tempPath = dev_tempPaths;
				dev_tempPaths = dev_paths;
				dev_paths = tempPath;
				ShadeableIntersection* tempInter = dev_tempIntersections;
				dev_tempIntersections = dev_intersections;
				dev_intersections = tempInter;
			}

		}
#endif



		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, num_paths, depth,
			dev_intersections, dev_paths, dev_materials
		);
		
		
		markPathSegment << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_bools, dev_nbools, dev_paths);
		int lastBool;
		int scanSum;
		hipMemcpy(&lastBool, dev_bools + pixelcount - 1, sizeof(int), hipMemcpyDeviceToHost);
		
		thrust::device_ptr<int> dv_in(dev_bools);
		thrust::device_ptr<int> dv_out(dev_scanBools);
		thrust::exclusive_scan(dv_in, dv_in + pixelcount, dv_out);
		hipMemcpy(&scanSum, dev_scanBools + pixelcount - 1, sizeof(int), hipMemcpyDeviceToHost);
		scanSum += lastBool;
		

		if (scanSum > 0) {
			thrust::device_ptr<int> dv_nin(dev_nbools);
			thrust::device_ptr<int> dv_nout(dev_scanNBools);
			thrust::exclusive_scan(dv_nin, dv_nin + pixelcount, dv_nout);
			pathSegmentScatter << <numBlocksPixels, blockSize1d >> > (pixelcount, scanSum, dev_tempPaths, dev_paths, dev_bools, dev_scanBools, dev_scanNBools);
			PathSegment* temp = dev_tempPaths;
			dev_tempPaths = dev_paths;
			dev_paths = temp;
		}

#ifdef DEBUG_OUTPUT
		std::cout << "iter-" << iter << ", depth-" << depth << ", paths: " << num_paths << " -> " << scanSum << std::endl;
#endif

		num_paths = scanSum;
		iterationComplete = num_paths <= 0 || depth >= traceDepth;
	
		if (guiData != NULL){ guiData->TracedDepth = depth; }
	}

	checkCUDAError("pathtrace before finalGather");

	// Assemble this iteration and apply it to the image
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

	checkCUDAError("pathtrace finalGather");

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	checkCUDAError("pathtrace sendImageToPBO");

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace final hipMemcpy");
}
