#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "main.h"

#define ERRORCHECK 1
#define DEBUG_OBJ_LOADER 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

#define CACHE_FIRST_BOUNCE 0
#define SORT_RAY_BY_MATERIAL 1

void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

// Predicate for stream compaction
struct isPathActive
{
	__host__ __device__ bool operator()(const int& stencil)
	{
		return stencil == 1;
	}
};

// sort the material 
struct materialSort
{
	__host__ __device__ bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b)
	{
		return a.materialId < b.materialId;
	}
};


__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;

static ShadeableIntersection* dev_intersections = NULL;
static PathSegment* dev_cache_paths = NULL;
static ShadeableIntersection* dev_first_bounce_intersections = NULL;
static int* dev_stencil = NULL;

// for bvh 
#if USE_BVH 1
static CompactBVH* dev_bvh = NULL;
#endif

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));

	// cache the path and first bounce intersection
	hipMalloc(&dev_cache_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_first_bounce_intersections, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_stencil, pixelcount * sizeof(int));

#if USE_BVH
	hipMalloc(&dev_bvh, scene->bvh.size() * sizeof(CompactBVH));
	hipMemcpy(dev_bvh, scene->bvh.data(), scene->bvh.size() * sizeof(CompactBVH), hipMemcpyHostToDevice);
#endif 
		
	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);

	hipFree(dev_intersections);
	hipFree(dev_first_bounce_intersections);
	hipFree(dev_stencil);

	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

#if CACHE_FIRST_BOUNCE
		float jitterX = 0.0f, jitterY = 0.0f;
#else
		// antialiasing by jittering the ray
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

		float jitterX = u01(rng), jitterY = u01(rng);
#endif 
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + jitterX)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + jitterY)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == TRIANGLE) {
				t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void computeIntersectionsBVH(
	int depth, int num_paths, PathSegment* pathSegments, Geom* geoms, 
	int geoms_size, ShadeableIntersection* intersections, CompactBVH* bvh, int bvh_size) {

	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths) {

		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// BVH traversal
		int stack[64];
		int stack_ptr = 0;
		stack[stack_ptr++] = 0;

		// bfs
		while (stack_ptr > 0) {
			int node_index = stack[--stack_ptr];

			if (node_index < bvh_size) {
				auto& boundingVol = bvh[node_index];

				if (intersectBVHNode(pathSegment.ray, boundingVol)) {
					if (boundingVol.geomStartIndex != -1) {   // hit the leaf node
						int start = boundingVol.geomStartIndex, end = boundingVol.geomEndIndex;

						for (int i = start; i < end; ++i) {
							auto& geom = geoms[i];

							if (geom.type == CUBE)
							{
								t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
							}
							else if (geom.type == SPHERE)
							{
								t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
							}
							else if (geom.type == TRIANGLE) {
								t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
							}

							// Compute the minimum t from the intersection tests to determine what
							// scene geometry object was hit first.
							if (t > 0.0f && t_min > t)
							{
								t_min = t;
								hit_geom_index = i;
								intersect_point = tmp_intersect;
								normal = tmp_normal;
							}
						}	
					}	
					else {
						// internal node, append the right child first,
						// and then left child
						if (node_index + boundingVol.rightChildOffset < bvh_size &&
							boundingVol.rightChildOffset > 0) {
							stack[stack_ptr++] = node_index + boundingVol.rightChildOffset;
						}

						if (node_index + 1 < bvh_size) {
							stack[stack_ptr++] = node_index + 1;
						}
					}
				}
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter, int num_paths, ShadeableIntersection* shadeableIntersections, 
	PathSegment* pathSegments, Material* materials) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		    // Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
		}
		else {
			pathSegments[idx].color = glm::vec3(0);
		}
	}
}

__global__ void shadeBSDFMaterial(
	int iter, int num_paths, ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments, Material* materials){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
			// Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				glm::vec3 intersectPoint = getPointOnRay(pathSegments[idx].ray, intersection.t);
				scatterRay(pathSegments[idx], intersectPoint, intersection.surfaceNormal,
					material, rng);
			}
		} 
		else {
			// If there was no intersection, color the ray black and terminate the ray.
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}


__global__ void computeStencil(int num_paths, PathSegment* paths, int* stencil) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx < num_paths) {
		stencil[idx] = (paths[idx].remainingBounces > 0) ? 1 : 0;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	printf("Iter: %d\n", iter);
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	if (!CACHE_FIRST_BOUNCE || iter == 1) {
		generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
		checkCUDAError("generate camera ray");

		// cache the paths
		hipMemcpy(dev_cache_paths, dev_paths, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
	}
	
	int depth = 0;

	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	bool iterationComplete = false;
	while (!iterationComplete) {
		// clean shading chunks
		// hipMemset(dev_intersections, 0, num_paths * sizeof(ShadeableIntersection));

		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		if (CACHE_FIRST_BOUNCE && depth == 0) {
			if (iter == 1) {
#if USE_BVH
				computeIntersectionsBVH << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections,
					dev_bvh, hst_scene->bvh.size());
				checkCUDAError("BVH computer intersection failed!");
#else
				computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections);
				checkCUDAError("computer intersection failed!");

#endif
				// cache the first bounce
				hipMemcpy(dev_first_bounce_intersections, dev_intersections,
					pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
			else {
				hipMemcpy(dev_paths, dev_cache_paths, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
				hipMemcpy(dev_intersections, dev_first_bounce_intersections,
					pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
		}
		else {
#if USE_BVH
			computeIntersectionsBVH << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections,
				dev_bvh, hst_scene->bvh.size());
			checkCUDAError("BVH computer intersection failed!");
#else
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections);
			checkCUDAError("computer intersection failed!");
#endif
		}

		hipDeviceSynchronize();
		depth++;

		/*printf("Depth: %d\n", depth);
		printf("Num of Paths: %d\n", num_paths);*/

#if DEBUG_OBJ_LOADER
		shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, num_paths, dev_intersections, dev_paths, dev_materials);
		iterationComplete = true;
#else

#if SORT_RAY_BY_MATERIAL
		// sort rays by material type 
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, materialSort());
#endif 

		// shade
		shadeBSDFMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, num_paths, dev_intersections, dev_paths, dev_materials);
		checkCUDAError("shade BDFS material failed!");

		computeStencil << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_paths, dev_stencil);

		// stream compaction
		auto new_end = thrust::stable_partition(thrust::device, dev_paths, dev_paths + num_paths, dev_stencil, isPathActive());
		num_paths = new_end - dev_paths;


		if (num_paths <= 0 || depth >= traceDepth) {
			// printf("End num path: %d\n", num_paths);
			iterationComplete = true;
		}
#endif
		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
