#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include "gpuScene.h"
#include "scene.h"
#include "rng.h"
#include "cudaTexture.h"

static constexpr int Compact_Threshold = 2073600;

struct CopyEndPaths 
{
	CPU_GPU bool operator() (const PathSegment& segment) {
		return segment.IsEnd();
	}
};

struct RemoveEndPaths 
{
	CPU_GPU bool operator() (const PathSegment& segment) {
		return segment.pixelIndex < 0 || segment.IsEnd();
	}
};

CPU_ONLY CudaPathTracer::~CudaPathTracer()
{
	// free ptr
	SafeCudaFree(dev_hdr_img);  // no-op if dev_image is null
	SafeCudaFree(dev_paths);
	SafeCudaFree(dev_end_paths);
	SafeCudaFree(dev_intersections);

	if (cuda_pbo_dest_resource)
	{
		UnRegisterPBO();
	}

	checkCUDAError("Free cuda pointers Error!");
}

GPU_ONLY float4 CudaTexture2D::Get(const float& x, const float& y) const
{
	return tex2D<float4>(m_TexObj, x, y);
}

CPU_GPU void writePixel(glm::vec3& hdr_pixel, uchar4& pixel)
{
	// tone mapping
	hdr_pixel = hdr_pixel / (1.f + hdr_pixel);

	// gammar correction
	hdr_pixel = glm::pow(hdr_pixel, glm::vec3(1.f / 2.2f));

	// map to [0, 255]
	hdr_pixel = glm::mix(glm::vec3(0.f), glm::vec3(255.f), hdr_pixel);
	
	hdr_pixel = glm::clamp(hdr_pixel, 0.f, 255.f);

	// write color
	pixel = { static_cast<unsigned char>(hdr_pixel.r), 
			  static_cast<unsigned char>(hdr_pixel.g), 
			  static_cast<unsigned char>(hdr_pixel.b), 
			  255 };
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, glm::vec3* image) 
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x >= resolution.x || y >= resolution.y) return;

	int index = (x + (y * resolution.x));
	glm::vec3 pix = image[index];

	writePixel(pix, pbo[index]);
}

static GuiDataContainer* guiData = nullptr;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

CPU_GPU Ray CastRay(const Camera& camera, const glm::vec2& p, const glm::vec2& rand_offset)
{
	glm::vec2 ndc = 2.f * p / glm::vec2(camera.resolution);
	ndc.x = ndc.x - 1.f;
	ndc.y = 1.f - ndc.y;

	float aspect = static_cast<float>(camera.resolution.x) / static_cast<float>(camera.resolution.y);

	// point in camera space
	float radian = glm::radians(camera.fovy * 0.5f);
	glm::vec3 p_camera = glm::vec3(
		ndc.x * glm::tan(radian) * aspect,
		ndc.y * glm::tan(radian),
		1.f
	);

	Ray ray(glm::vec3(0), p_camera);

	// len camera
	glm::vec2 p_len = camera.lenRadius * SquareToDiskConcentric(rand_offset);
	glm::vec3 p_focal = camera.focalDistance * p_camera;
	ray.origin.x = p_len.x;
	ray.origin.y = p_len.y;
	ray.direction = glm::normalize(p_focal - ray.origin);

	// transform to world space
	ray.origin = camera.position + ray.origin.x * camera.right + ray.origin.y * camera.up;
	ray.direction = glm::normalize(
		ray.direction.z * camera.forward +
		ray.direction.y * camera.up +
		ray.direction.x * camera.right
	);

	return ray;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) 
	{
		int index = x + (y * cam.resolution.x);
		PathSegment segment;
		segment.Reset();

		CudaRNG rng(iter, index, 0);

		segment.ray = CastRay(cam, { x + rng.rand(), y + rng.rand() }, { rng.rand(), rng.rand() });
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;

		pathSegments[index] = segment;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(int num_paths, PathSegment* pathSegments, ShadeableIntersection* intersections, GPUScene scene)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= num_paths) return;
	
	PathSegment segment = pathSegments[index];
	ShadeableIntersection& shadeable_intersection = intersections[index];
	shadeable_intersection.Reset();

	if (segment.remainingBounces <= 0) return;

	Intersection intersection = scene.SceneIntersection(segment.ray, threadIdx.x);
	if (intersection.shapeId >= 0)
	{
		ShadeableIntersection shadeable;
		shadeable.t = intersection.t;
		shadeable.position = segment.ray * intersection.t;
		glm::ivec3 n_id = scene.dev_triangles[intersection.shapeId].n_id;
		glm::ivec3 uv_id = scene.dev_triangles[intersection.shapeId].uv_id;

		shadeable.normal = BarycentricInterpolation<glm::vec3>(scene.dev_normals[n_id.x],
															   scene.dev_normals[n_id.y],
															   scene.dev_normals[n_id.z], intersection.uv);;
		shadeable.normal = glm::normalize(shadeable.normal);
		shadeable.uv = BarycentricInterpolation<glm::vec2>(scene.dev_uvs[uv_id.x], 
														   scene.dev_uvs[uv_id.y], 
														   scene.dev_uvs[uv_id.z], intersection.uv);;
		shadeable.materialId = intersection.materialId;

		shadeable_intersection = shadeable;
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(float u, int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= nPaths) return;

	PathSegment segment = iterationPaths[index];

	glm::vec3 pre_color = image[segment.pixelIndex];
	glm::vec3 new_color = glm::mix(pre_color, segment.radiance, u);

	image[segment.pixelIndex] = new_color;
}

// Naive BSDF sample only
__global__ void KernelNaiveGI(const int iteration, const int num_paths, const int num_materials,
							ShadeableIntersection* shadeableIntersections,
							PathSegment* pathSegments,
							const Material* materials, EnvironmentMap env_map, 
							const UniformMaterialData u_data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;
	__shared__ Material shared_materials[128];
	if (blockDim.x >= num_materials)
	{
		if (threadIdx.x < 128 && threadIdx.x < num_materials)
		{
			shared_materials[threadIdx.x] = materials[threadIdx.x];
		}
		__syncthreads();
	}

	PathSegment segment = pathSegments[idx];
	if (segment.IsEnd()) return;
	ShadeableIntersection intersection = shadeableIntersections[idx];
	
	if (intersection.materialId >= 0)
	{
		if (segment.mediaId >= 0)
		{
			CudaRNG rng(iteration, idx, segment.remainingBounces);

			const float distance = -glm::log(rng.rand()) / u_data.ss_scatter_coeffi;
			if (distance < 1000.f)
			{
				if (distance < intersection.t)
				{
					Ray ray(segment.ray.origin + segment.ray.direction * distance, SquareToSphereUniform({ rng.rand(), rng.rand() }));
					pathSegments[idx].ray = ray;

					const float weight = glm::exp(-u_data.ss_scatter_coeffi * distance);
					const float pdf = Inv4Pi;
					const glm::vec3 transmission = glm::exp(-u_data.ss_absorption_coeffi * distance);
					pathSegments[idx].throughput *= transmission * glm::max(weight, 0.1f);
					return;
				}
				else
				{
					pathSegments[idx].throughput *= glm::exp(-u_data.ss_absorption_coeffi * intersection.t);
					const glm::vec3 transmission = glm::exp(-u_data.ss_absorption_coeffi * intersection.t);
					pathSegments[idx].throughput *= transmission;
				}
			}
		}
		
		Material material;
		if (blockDim.x >= num_materials)
		{
			material = materials[intersection.materialId];
		}
		else
		{
			Material material = shared_materials[intersection.materialId];
		}

		if (intersection.materialId == 0)
		{
			material.type = u_data.type;
			material.eta = u_data.eta;
			material.data.values.albedo = u_data.albedo;
			material.data.values.metallic = u_data.metallic;
			material.data.values.roughness = u_data.roughness;
		}

		if (material.emittance > 0.f) 
		{
			glm::vec3 final_throughput = segment.throughput * material.emittance;
			pathSegments[idx].radiance = final_throughput;
			pathSegments[idx].Terminate();
		}
		else
		{	
			material.GetNormal(intersection.uv, intersection.normal);
			CudaRNG rng(iteration, idx, segment.remainingBounces);
			BSDFSample bsdf_sample;
			bsdf_sample.wiW = -segment.ray.direction;
			if(SampleBSDF::Sample(material, intersection, rng, bsdf_sample))
			{
				// generate new ray
				pathSegments[idx].ray = Ray::SpawnRay(intersection.position, bsdf_sample.wiW);
				pathSegments[idx].throughput *= bsdf_sample.f * glm::abs(glm::dot(bsdf_sample.wiW, intersection.normal)) / bsdf_sample.pdf;
				--pathSegments[idx].remainingBounces;

				if (MaterialType::SubsurfaceScattering == material.type)
				{
					pathSegments[idx].mediaId = segment.mediaId >= 0 ? -1: 0;
				}
			}
			else
			{
				pathSegments[idx].remainingBounces = 0;
				return;
			}
		}
	}
	else
	{
		if (env_map.Valid())
		{
			pathSegments[idx].radiance = segment.throughput * glm::clamp(env_map.Get(segment.ray.direction), 0.f, 200.f);
		}
		pathSegments[idx].Terminate();
	}
}

__global__ void KernelDisplayNormal(const int iteration, const int num_paths, const int num_materials,
										ShadeableIntersection* shadeableIntersections,
										PathSegment* pathSegments,
										const Material* materials)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;

	PathSegment segment = pathSegments[idx];
	if (segment.IsEnd()) return;
	ShadeableIntersection intersection = shadeableIntersections[idx];

	if (intersection.materialId >= 0)
	{
		Material material = materials[intersection.materialId];
		material.GetNormal(intersection.uv, intersection.normal);
		pathSegments[idx].radiance = intersection.normal * 0.5f + 0.5f;
	}
	pathSegments[idx].Terminate();
	return;
}

CPU_ONLY void CudaPathTracer::Resize(const int& w, const int& h)
{
	resolution.x = w;
	resolution.y = h;

	SafeCudaFree(dev_hdr_img);  // no-op if dev_image is null
	SafeCudaFree(dev_paths);
	SafeCudaFree(dev_end_paths);
	SafeCudaFree(dev_intersections);

	if (cuda_pbo_dest_resource)
	{
		UnRegisterPBO();
	}
	const int pixelcount = resolution.x * resolution.y;
	
	checkCUDAError("Get PBO pointer Error");

	hipMalloc(&dev_hdr_img, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_hdr_img, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_end_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	thrust_dev_paths_begin = thrust::device_ptr<PathSegment>(dev_paths);
	thrust_dev_end_paths_bgein = thrust::device_ptr<PathSegment>(dev_end_paths);
}

void CudaPathTracer::Init(Scene* scene)
{
	m_Iteration = 0;

	const Camera& cam = scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;
	
	resolution = cam.resolution;

	hipMalloc(&dev_hdr_img, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_hdr_img, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_end_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	thrust_dev_paths_begin = thrust::device_ptr<PathSegment>(dev_paths);
	thrust_dev_end_paths_bgein = thrust::device_ptr<PathSegment>(dev_end_paths);

	checkCUDAError("Create device image error");
}

CPU_ONLY void CudaPathTracer::GetImage(uchar4* host_image)
{
	//Retrieve image from GPU
	hipMemcpy(host_image, dev_img, resolution.x * resolution.y * sizeof(uchar4), hipMemcpyDeviceToHost);
}

CPU_ONLY void CudaPathTracer::RegisterPBO(unsigned int pbo)
{
	hipGraphicsGLRegisterBuffer(&cuda_pbo_dest_resource, pbo, cudaGraphicsMapFlagsNone);
	size_t byte_count = resolution.x * resolution.y * 4 * sizeof(uchar4);
	hipGraphicsMapResources(1, &cuda_pbo_dest_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&dev_img, &byte_count, cuda_pbo_dest_resource);
	checkCUDAError("Get PBO pointer Error");
}

CPU_ONLY void CudaPathTracer::Render(GPUScene& scene, 
									 const Camera& camera,
									 const UniformMaterialData& data)
{
	const int pixelcount = resolution.x * resolution.y;

	const int& max_depth = camera.path_depth;
	// TODO: might change to dynamic block size
	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (camera, m_Iteration, max_depth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;

	int num_paths = pixelcount;

	thrust::device_ptr<PathSegment> thrust_end_paths_end = thrust_dev_end_paths_bgein;
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (depth < max_depth && num_paths > 0)
	{
		depth++;

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_paths, dev_intersections, scene);
		checkCUDAError("Intersection Error");
		hipDeviceSynchronize();
#if DebugNormal
		KernelDisplayNormal << <numblocksPathSegmentTracing, blockSize1d >> > (m_Iteration, num_paths, scene.material_count,
																				dev_intersections, dev_paths, scene.dev_materials);
#else
		KernelNaiveGI<<<numblocksPathSegmentTracing, blockSize1d >>>(m_Iteration, num_paths, scene.material_count,
																	 dev_intersections, dev_paths, scene.dev_materials, scene.env_map, data);
		checkCUDAError("NaiveGI Error");
#endif
		hipDeviceSynchronize();
		if (pixelcount >= Compact_Threshold)
		{
			// remove terminated segments
			thrust_end_paths_end = thrust::copy_if(thrust_dev_paths_begin, thrust_dev_paths_begin + num_paths, thrust_end_paths_end, CopyEndPaths());
			auto remove_ptr = thrust::remove_if(thrust_dev_paths_begin, thrust_dev_paths_begin + num_paths, CopyEndPaths());

			num_paths = remove_ptr - thrust_dev_paths_begin;
		}
	}

	// Assemble this iteration and apply it to the image
	float u = 1.f / static_cast<float>(m_Iteration + 1); // used for interpolation between last frame and this frame
	if (pixelcount >= Compact_Threshold)
	{
		int num_end_paths = thrust_end_paths_end - thrust_dev_end_paths_bgein;
		dim3 numBlocksPixels = (num_end_paths + blockSize1d - 1) / blockSize1d;

		finalGather << <numBlocksPixels, blockSize1d >> > (u, num_end_paths, dev_hdr_img, dev_end_paths);
	}
	else
	{
		dim3 numBlocksPixels = (num_paths + blockSize1d - 1) / blockSize1d;

		finalGather << <numBlocksPixels, blockSize1d >> > (u, num_paths, dev_hdr_img, dev_paths);
	}
	checkCUDAError("Final Gather failed");
	hipDeviceSynchronize();
	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (dev_img, camera.resolution, dev_hdr_img);

	checkCUDAError("pathtrace");
	++m_Iteration;
}