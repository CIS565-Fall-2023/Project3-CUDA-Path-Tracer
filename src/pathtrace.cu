#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include "common.h"
#include "sampler.h"

struct CompactTerminatedPaths {
	CPU_GPU bool operator() (const PathSegment& segment) {
		return !(segment.pixelIndex >= 0 && segment.IsEnd());
	}
};

struct RemoveInvalidPaths {
	CPU_GPU bool operator() (const PathSegment& segment) {
		return segment.pixelIndex < 0 || segment.IsEnd();
	}
};

CPU_ONLY CudaPathTracer::~CudaPathTracer()
{
	// free ptr
	SafeCudaFree(dev_hdr_img);  // no-op if dev_image is null
	SafeCudaFree(dev_paths);
	SafeCudaFree(dev_geoms);
	SafeCudaFree(dev_materials);
	SafeCudaFree(dev_intersections);

	if (cuda_pbo_dest_resource)
	{
		UnRegisterPBO();
	}

	checkCUDAError("CudaPathTracer delete Error!");
}

CPU_ONLY GPU_ONLY thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

CPU_GPU void writePixel(glm::vec3& hdr_pixel, uchar4& pixel)
{
	// tone mapping
	hdr_pixel = hdr_pixel / (1.f + hdr_pixel);

	// gammar correction
	hdr_pixel = glm::pow(hdr_pixel, glm::vec3(1.f / 2.2f));

	// map to [0, 255]
	hdr_pixel = glm::mix(glm::vec3(0.f), glm::vec3(255.f), hdr_pixel);
	
	// write color
	pixel = { static_cast<unsigned char>(hdr_pixel.r), 
			  static_cast<unsigned char>(hdr_pixel.g), 
			  static_cast<unsigned char>(hdr_pixel.b), 
			  255 };
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, glm::vec3* image) 
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x >= resolution.x || y >= resolution.y) return;

	int index = x + (y * resolution.x);
	glm::vec3 pix = image[index];

	writePixel(pix, pbo[index]);
}

static Scene* hst_scene = nullptr;
static GuiDataContainer* guiData = nullptr;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];
		segment.ray = cam.CastRay({x, y});

		segment.throughput = glm::vec3(1.0f);
		segment.radiance = glm::vec3(0.0f);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];
		if (pathSegment.IsEnd()) 
		{
			intersections[path_index].t = -1.0f;
			return;
		}
		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].surfacePosition = intersect_point;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].throughput *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].throughput *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].throughput *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].throughput = glm::vec3(0.0f);
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(float u, int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= nPaths) return;

	PathSegment iterationPath = iterationPaths[index];
	glm::vec3 pre_color = image[iterationPath.pixelIndex];
	glm::vec3 new_color = glm::mix(pre_color, iterationPath.radiance, u);

	image[iterationPath.pixelIndex] = new_color;
}

// Naive BSDF sample only
__global__ void KernelNaiveGI(int iteration, int num_paths, 
							ShadeableIntersection* shadeableIntersections,
							PathSegment* pathSegments,
							Material* materials)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;

	ShadeableIntersection intersection = shadeableIntersections[idx];
	PathSegment segment = pathSegments[idx];

	if (intersection.t > 0.f && !segment.IsEnd())
	{
		Material material = materials[intersection.materialId];
		glm::vec3 materialColor = material.color;
		//pathSegments[idx].radiance = intersection.surfaceNormal * 0.5f + 0.5f;
		if (material.emittance > 0.0f) {
			glm::vec3 final_throughput = pathSegments[idx].throughput * material.emittance;
			pathSegments[idx].radiance = final_throughput;
			pathSegments[idx].Terminate();
		}
		else
		{		
			glm::vec3 wo = WorldToLocal(intersection.surfaceNormal) * -segment.ray.direction;
			if (wo.z < 0.f)
			{
				pathSegments[idx].Terminate();
				return;
			}

			thrust::default_random_engine rng = makeSeededRandomEngine(iteration, idx, 0);
			thrust::uniform_real_distribution<float> u01(0.f, 1.f);

			// naive diffuse surface
			glm::vec3 wi = SquareToHemisphereCosine({ u01(rng), u01(rng) });
			glm::vec3 wiW = glm::normalize(LocalToWorld(intersection.surfaceNormal) * wi);
		
			float pdf = SquareToHemisphereCosinePDF(wi);
		
			// generate new ray
			pathSegments[idx].ray = Ray::SpawnRay(intersection.surfacePosition, wiW);
			pathSegments[idx].throughput *= materialColor * InvPi;// *wi.z / pdf;
		}
	}
	else
	{
		pathSegments[idx].Terminate();
	}
}

CPU_ONLY void CudaPathTracer::Init(Scene* scene)
{
	m_Iteration = 0;
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_hdr_img, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_hdr_img, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_terminated_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need

	thrust_dev_paths = thrust::device_ptr<PathSegment>(dev_paths);
	thrust_dev_terminated_paths = thrust::device_ptr<PathSegment>(dev_terminated_paths);

	checkCUDAError("pathtraceInit");
}

CPU_ONLY void CudaPathTracer::GetImage(uchar4* host_image)
{
	const Camera& cam = hst_scene->state.camera;
	// Retrieve image from GPU
	hipMemcpy(host_image, dev_img, cam.resolution.x * cam.resolution.y * sizeof(uchar4), hipMemcpyDeviceToHost);
}

CPU_ONLY void CudaPathTracer::RegisterPBO(unsigned int pbo)
{
	hipGraphicsGLRegisterBuffer(&cuda_pbo_dest_resource, pbo, cudaGraphicsMapFlagsNone);
	size_t byte_count = resolution.x * resolution.y * 4 * sizeof(uchar4);
	hipGraphicsMapResources(1, &cuda_pbo_dest_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&dev_img, &byte_count, cuda_pbo_dest_resource);
	checkCUDAError("Get PBO pointer Error");
}

CPU_ONLY void CudaPathTracer::Render()
{
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// TODO: might change to dynamic block size
	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, m_Iteration, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (depth < 5 && num_paths > 0) {
		depth++;

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
	  // evaluating the BSDF.
	  // Start off with just a big kernel that handles all the different
	  // materials you have in the scenefile.
	  // TODO: compare between directly shading the path segments and shading
	  // path segments that have been reshuffled to be contiguous in memory.

		//shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
		//	iter,
		//	num_paths,
		//	dev_intersections,
		//	dev_paths,
		//	dev_materials
		//	);
		//iterationComplete = true; // TODO: should be based off stream compaction results.
		KernelNaiveGI << <numblocksPathSegmentTracing, blockSize1d >> > (m_Iteration, num_paths,
			dev_intersections, dev_paths, dev_materials);

		if (guiData != nullptr)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;

	float u = 1.f / static_cast<float>(m_Iteration + 1); // used for interpolation between last frame and this frame

	finalGather << <numBlocksPixels, blockSize1d >> > (u, num_paths, dev_hdr_img, dev_paths);
	checkCUDAError("Final Gather failed");
	hipDeviceSynchronize();
	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (dev_img, cam.resolution, dev_hdr_img);

	checkCUDAError("pathtrace");
	++m_Iteration;
}