#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <numeric>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static PathSegment* dev_tempPaths = NULL;
static PathSegment* dev_pathsBuffer = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static ShadeableIntersection* dev_tempIntersections = NULL;
static ShadeableIntersection* dev_intersectionsBuffer = NULL;
static int* dev_bools = NULL;
static int* dev_nbools = NULL;
static int* dev_scanBools = NULL;
static int* dev_scanNBools = NULL;
static Triangle* dev_tris = NULL;
static BoundingBox* dev_bvh = NULL;


void printArr(int n, int* odata, int* dev_odata) {
	hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy dev_odata -> odata for printArr failed!");
	for (int i = 0; i <= n / 10; i++) {
		for (int j = 0; j < 10 && j < n - 10 * i; j++) {
			std::cout << odata[i * 10 + j] << "  ";
		}std::cout << std::endl;
	}std::cout << std::endl << std::endl;
}

void printArr(int begin, int n, int* dev_odata) {
	int o[10];
	for (int i = 0; i <= n / 10; i++) {
		hipMemcpy(o, dev_odata + begin + 10 * i, sizeof(int) * 10, hipMemcpyDeviceToHost);
		if (o[0]+o[1]+o[2]+o[3]+o[4]+o[5]+o[6]+o[7]+o[8]+o[9] == -10) {
			continue;
		}
		checkCUDAError("hipMemcpy dev_odata -> odata for printArr failed!");
		for (int j = 0; j < 10 && j < n - 10 * i; j++) {
			std::cout << o[j] << "  ";
		}std::cout << std::endl;
	}std::cout << std::endl << std::endl;
}

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_tempPaths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_tris, scene->tris.size() * sizeof(Triangle));
	hipMemcpy(dev_tris, scene->tris.data(), scene->tris.size() * sizeof(Triangle), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	hipMalloc(&dev_tempIntersections, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_bools, pixelcount * sizeof(int));
	hipMalloc(&dev_nbools, pixelcount * sizeof(int));
	hipMalloc(&dev_scanBools, pixelcount * sizeof(int));
	hipMalloc(&dev_scanNBools, pixelcount * sizeof(int));

#ifdef CACHE_FIRST_BOUNCE
	hipMalloc(&dev_intersectionsBuffer, pixelcount * sizeof(ShadeableIntersection));
	hipMalloc(&dev_pathsBuffer, pixelcount * sizeof(PathSegment));
#endif

#ifdef USING_BVH
	hipMalloc(&dev_bvh, scene->bvh.size() * sizeof(BoundingBox));
	hipMemcpy(dev_bvh, scene->bvh.data(), scene->bvh.size() * sizeof(BoundingBox), hipMemcpyHostToDevice);
#endif

	checkCUDAError("pathtraceInit");	
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_tris);
	hipFree(dev_materials);
	hipFree(dev_intersections);

	hipFree(dev_bools);
	hipFree(dev_nbools);
	hipFree(dev_scanBools);
	hipFree(dev_scanNBools);
	hipFree(dev_tempPaths);
	hipFree(dev_tempIntersections);
#ifdef CACHE_FIRST_BOUNCE
	hipFree(dev_intersectionsBuffer);
	hipFree(dev_pathsBuffer);
#endif
#ifdef USING_BVH
	hipFree(dev_bvh);
#endif
	checkCUDAError("pathtraceFree");
}



/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/

__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;


	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		thrust::random::default_random_engine rng = makeSeededRandomEngine(iter, x, y);
#ifdef JITTER_RAY
		thrust::uniform_real_distribution<float> u1(-1, 1);
		thrust::uniform_real_distribution<float> u02PI(0, TWO_PI);
		float rz = sqrt(u1(rng));
		rz = ((rz > 0) ? 1 : -1) * sqrt(abs(rz));
#endif

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
#ifdef JITTER_RAY
			+ glm::normalize(glm::vec3(__sinf(u02PI(rng)), __cosf(u02PI(rng)), rz)) * JITTER_RATIO
#endif
		);


		// Depth of field
#ifdef DEPTH_OF_FIELD
		thrust::uniform_real_distribution<float> u01(-1, 1);
		glm::vec2 sample = glm::vec2(u01(rng), u01(rng));
		glm::vec2 sampleDisk;
		if (length(sample) < 1e-5) { sampleDisk = glm::vec2(0, 0); }
		else {
			float theta, r;
			if (std::abs(sample.x) > std::abs(sample.y)) {
				r = sample.x;
				theta = (PI / 4.0) * (sample.y / sample.x);
			}
			else {
				r = sample.y;
				theta = (PI / 2.0) - (PI / 4.0) * (sample.x / sample.y);
			}
			sampleDisk = r * glm::vec2(std::cos(theta), std::sin(theta));
		}

		glm::vec2 pLens = DOF_LENS_RADIUS * sampleDisk;

		float ft = glm::abs(DOF_FOCAL_DISTANCE / segment.ray.direction.z);
		glm::vec3 pFocus = segment.ray.origin + ft * segment.ray.direction;

		segment.ray.origin += cam.right * pLens.x + cam.up * pLens.y;
		segment.ray.direction = glm::normalize(pFocus - segment.ray.origin);
#endif


		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
		segment.refractionBefore = false;
	}
}


// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersectionsNaive(
	int depth
	, int num_paths
	, const PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, Triangle* tris
	, int tris_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_index = -1;
		bool hit_geom = true;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		for (int i = 0; i < tris_size; i++) {
			Triangle& tri = tris[i];
			t = triangleIntersectionTest(tri, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_index = i;
				hit_geom = false;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = hit_geom ? geoms[hit_index].materialid : tris[hit_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}




__global__ void computeIntersectionsBVH(
	int depth
	, int num_paths
	, const PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, Triangle* tris
	, int tris_size
	, BoundingBox* bvh
	, int bvh_size
	, ShadeableIntersection* intersections
) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = 1e5;
		int hit_index = -1;
		bool hit_geom = true;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}


		// __shared__ int arr[BLOCK_SIZE_1D][BVH_GPU_STACK_SIZE];
		int arr[BVH_GPU_STACK_SIZE];

		// int arr[BVH_GPU_STACK_SIZE];
		int sign = 0;
		arr[0] = 0;

		float o[3] = { pathSegment.ray.origin.x, pathSegment.ray.origin.y, pathSegment.ray.origin.z };

		glm::vec3 inv_dir = 1.0f / pathSegment.ray.direction;

		while (sign >= 0) {
			
			BoundingBox& bbox = bvh[arr[sign]];
			int beginId = bbox.beginTriId;
			sign--;


			float bbox_min_t = 1e-5;
			float bboX_max_t = 1e5;

			for (int a = 0; a < 3; a++) {

				float t0 = (bbox.min[a] - o[a]) * inv_dir[a];
				float t1 = (bbox.max[a] - o[a]) * inv_dir[a];

				bbox_min_t = fmax(min(t0, t1), bbox_min_t);
				bboX_max_t = fmin(max(t0, t1), bboX_max_t);
			}

			if(bboX_max_t >= bbox_min_t && bboX_max_t > 0.0f && bbox_min_t < t_min)
			{
				// reach leaf node of bvh
				if (beginId >= 0) {

					// TriangleArray& triIndices = tri_arr[taid];
					// #pragma unroll
					for (int j = beginId; j < bbox.triNum + beginId; j++) {

						t = triangleIntersectionTest(tris[j], pathSegment.ray, tmp_intersect, tmp_normal, outside);
						if (t > 0.0f && t_min > t)
						{
							t_min = t;
							hit_index = j;
							hit_geom = false;
							intersect_point = tmp_intersect;
							normal = tmp_normal;
						}
					}
				}
				// keep searching
				else if (sign + 2 < BVH_GPU_STACK_SIZE) {
					arr[sign + 1] = bbox.leftId;
					arr[sign + 2] = bbox.rightId;
					sign += 2;
				}
			}
		}



		if (hit_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = hit_geom ? geoms[hit_index].materialid : tris[hit_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}



// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
		}
		else {
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

__global__ void shadeMaterial(
	int iter
	, int num_paths
	, int depth
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths && pathSegments[idx].remainingBounces > 0)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
			// Set up the RNG
			// LOOK: this is how you use thrust's RNG! Please look at
			// makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
			// thrust::uniform_real_distribution<float> u01(0, 1); // u01(rng) to get random (0, 1)

			Material material = materials[intersection.materialId];

			glm::vec3 intersect = pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * intersection.t;
			scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = -1;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color * (float)abs(iterationPath.remainingBounces);		
	}
}

__global__ void markPathSegment(int nPaths, int* bools, int* nbools, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		bool b = iterationPaths[index].remainingBounces > 0;
		bools[index] = b;
		nbools[index] = !b;
	}
}

// for path termination
__global__ void pathSegmentScatter(int n, int scanSum, PathSegment* odata, const PathSegment* idata, const int* bools, const int* indicesPos, const int* indicesNeg) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		if (bools[index] > 0) {
			odata[indicesPos[index]] = idata[index];
		}
		else {
			odata[indicesNeg[index] + scanSum] = idata[index];
		}
	}
}

// for material sort
__global__ void kernMapMatBitToBoolean(int n, int i, int* bools, int* ebools, const ShadeableIntersection* idata) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		bools[index] = ((idata[index].materialId >> i) & 1);
		ebools[index] = !((idata[index].materialId >> i) & 1);
	}
}

// for path termination
__global__ void pathSegmentAndIntersectionScatter(
	int n, int negCount,
	PathSegment* opaths, const PathSegment* ipaths,
	ShadeableIntersection* ointers, const ShadeableIntersection* iinters,
	const int* bools, const int* indicesPos, const int* indicesNeg) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		if (bools[index] > 0) {
			opaths[indicesPos[index] + negCount] = ipaths[index];
			ointers[indicesPos[index] + negCount] = iinters[index];
		}
		else {
			opaths[indicesNeg[index]] = ipaths[index];
			ointers[indicesNeg[index]] = iinters[index];
		}
	}
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
 ///////////////////////////////////////////////////////////////////////////

void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = BLOCK_SIZE_1D;
	int depth = 0;
	int num_paths = pixelcount;


#ifdef CACHE_FIRST_BOUNCE
	if (iter == 1) {
		generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
		checkCUDAError("generate camera ray");
		hipMemcpy(dev_pathsBuffer, dev_paths, sizeof(PathSegment) * pixelcount, hipMemcpyDeviceToDevice);
		checkCUDAError("save dev_pathsBuffer");
	}
	else {
		hipMemcpy(dev_paths, dev_pathsBuffer, sizeof(PathSegment) * pixelcount, hipMemcpyDeviceToDevice);
		checkCUDAError("load dev_pathsBuffer");
	}
#else
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");
#endif


	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	int mat_num = hst_scene->materials.size();

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {
		
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		checkCUDAError("hipMemset dev_intersections");

#ifdef CACHE_FIRST_BOUNCE
		if (iter > 1 && depth == 0) {
			hipMemcpy(dev_intersections, dev_intersectionsBuffer, sizeof(ShadeableIntersection) * pixelcount, hipMemcpyDeviceToDevice);
			checkCUDAError("load dev_intersectionsBuffer");
		}
		else {
#endif
			// tracing
#ifdef USING_BVH
			if (hst_scene->bvh.size() > 0) {
				computeIntersectionsBVH << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth, num_paths, dev_paths,
					dev_geoms, hst_scene->geoms.size(),
					dev_tris, hst_scene->tris.size(),
					dev_bvh, hst_scene->bvh.size(),
					dev_intersections);
				checkCUDAError("tcomputeIntersectionsBVH");
			} else {
#endif
				computeIntersectionsNaive << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth, num_paths, dev_paths,
					dev_geoms, hst_scene->geoms.size(),
					dev_tris, hst_scene->tris.size(),
					dev_intersections);
#ifdef USING_BVH
			}
#endif


			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
#ifdef CACHE_FIRST_BOUNCE
			if (iter == 1 && depth == 0) {
				hipMemcpy(dev_intersectionsBuffer, dev_intersections, sizeof(ShadeableIntersection) * pixelcount, hipMemcpyDeviceToDevice);
				checkCUDAError("save dev_intersectionsBuffer");
			}
		}
#endif

		depth++;

		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.


		// Sort All Materials
#ifdef MATERIAL_SORT
		if (mat_num > 1) {
			int log2Ceil = 1;
			int product = 1;
			while (product < mat_num) {
				product *= 2;
				log2Ceil++;
			}
			for (int i = 0; i < log2Ceil; i++) {
				kernMapMatBitToBoolean << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, i, dev_bools, dev_nbools, dev_intersections);
				thrust::device_ptr<int> dv_sort_in(dev_bools);
				thrust::device_ptr<int> dv_sort_out(dev_scanBools);
				thrust::exclusive_scan(dv_sort_in, dv_sort_in + num_paths, dv_sort_out);
				thrust::device_ptr<int> dv_sort_nin(dev_nbools);
				thrust::device_ptr<int> dv_sort_nout(dev_scanNBools);
				thrust::exclusive_scan(dv_sort_nin, dv_sort_nin + num_paths, dv_sort_nout);

				int neg_count = -1;
				hipMemcpy(&neg_count, dev_scanNBools + num_paths - 1, sizeof(int), hipMemcpyDeviceToHost);
				int last_bool = -1;
				hipMemcpy(&last_bool, dev_bools + num_paths - 1, sizeof(int), hipMemcpyDeviceToHost);
				neg_count += (last_bool == 0);

				pathSegmentAndIntersectionScatter << < numblocksPathSegmentTracing, blockSize1d >> > (
					num_paths, neg_count,
					dev_tempPaths, dev_paths,
					dev_tempIntersections, dev_intersections,
					dev_bools, dev_scanBools, dev_scanNBools);

				PathSegment* tempPath = dev_tempPaths;
				dev_tempPaths = dev_paths;
				dev_paths = tempPath;
				ShadeableIntersection* tempInter = dev_tempIntersections;
				dev_tempIntersections = dev_intersections;
				dev_intersections = tempInter;
			}

		}
#endif



		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, num_paths, depth,
			dev_intersections, dev_paths, dev_materials
		);
		
		
		markPathSegment << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_bools, dev_nbools, dev_paths);
		int lastBool;
		int scanSum;
		hipMemcpy(&lastBool, dev_bools + pixelcount - 1, sizeof(int), hipMemcpyDeviceToHost);
		
		thrust::device_ptr<int> dv_in(dev_bools);
		thrust::device_ptr<int> dv_out(dev_scanBools);
		thrust::exclusive_scan(dv_in, dv_in + pixelcount, dv_out);
		hipMemcpy(&scanSum, dev_scanBools + pixelcount - 1, sizeof(int), hipMemcpyDeviceToHost);
		scanSum += lastBool;
		

		if (scanSum > 0) {
			thrust::device_ptr<int> dv_nin(dev_nbools);
			thrust::device_ptr<int> dv_nout(dev_scanNBools);
			thrust::exclusive_scan(dv_nin, dv_nin + pixelcount, dv_nout);
			pathSegmentScatter << <numBlocksPixels, blockSize1d >> > (pixelcount, scanSum, dev_tempPaths, dev_paths, dev_bools, dev_scanBools, dev_scanNBools);
			PathSegment* temp = dev_tempPaths;
			dev_tempPaths = dev_paths;
			dev_paths = temp;
		}

#ifdef DEBUG_OUTPUT
		std::cout << "iter-" << iter << ", depth-" << depth << ", paths: " << num_paths << " -> " << scanSum << std::endl;
#endif

		num_paths = scanSum;
		iterationComplete = num_paths <= 0 || depth >= traceDepth;
	
		if (guiData != NULL){ guiData->TracedDepth = depth; }
	}

	checkCUDAError("pathtrace before finalGather");

	// Assemble this iteration and apply it to the image
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

	checkCUDAError("pathtrace finalGather");

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	checkCUDAError("pathtrace sendImageToPBO");

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace final hipMemcpy");
}
