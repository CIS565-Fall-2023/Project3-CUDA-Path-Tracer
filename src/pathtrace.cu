#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include "common.h"
#include "sampler.h"
#include "gpuScene.h"

struct CompactTerminatedPaths {
	CPU_GPU bool operator() (const PathSegment& segment) {
		return !(segment.pixelIndex >= 0 && segment.IsEnd());
	}
};

struct RemoveInvalidPaths {
	CPU_GPU bool operator() (const PathSegment& segment) {
		return segment.pixelIndex < 0 || segment.IsEnd();
	}
};

CPU_ONLY CudaPathTracer::~CudaPathTracer()
{
	// free ptr
	SafeCudaFree(dev_hdr_img);  // no-op if dev_image is null
	SafeCudaFree(dev_paths);
	SafeCudaFree(dev_geoms);
	SafeCudaFree(dev_materials);
	SafeCudaFree(dev_intersections);

	if (cuda_pbo_dest_resource)
	{
		UnRegisterPBO();
	}

	checkCUDAError("CudaPathTracer delete Error!");
}

CPU_ONLY GPU_ONLY thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

CPU_GPU void writePixel(glm::vec3& hdr_pixel, uchar4& pixel)
{
	// tone mapping
	hdr_pixel = hdr_pixel / (1.f + hdr_pixel);

	// gammar correction
	hdr_pixel = glm::pow(hdr_pixel, glm::vec3(1.f / 2.2f));

	// map to [0, 255]
	hdr_pixel = glm::mix(glm::vec3(0.f), glm::vec3(255.f), hdr_pixel);
	
	// write color
	pixel = { static_cast<unsigned char>(hdr_pixel.r), 
			  static_cast<unsigned char>(hdr_pixel.g), 
			  static_cast<unsigned char>(hdr_pixel.b), 
			  255 };
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, glm::vec3* image) 
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x >= resolution.x || y >= resolution.y) return;

	int index = x + (y * resolution.x);
	glm::vec3 pix = image[index];

	int write_index = (resolution.x - x - 1) + (y * resolution.x);
	writePixel(pix, pbo[write_index]);
}

static Scene* hst_scene = nullptr;
static GuiDataContainer* guiData = nullptr;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) 
	{
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];
		segment.Reset();
		segment.ray = cam.CastRay({x, y});

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(int num_paths, PathSegment* pathSegments, ShadeableIntersection* intersections, GPUScene scene)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= num_paths) return;
	
	PathSegment segment = pathSegments[index];
	ShadeableIntersection& shadeable_intersection = intersections[index];
	shadeable_intersection.Reset();

	if (segment.IsEnd()) return;

	Intersection intersection = scene.SceneIntersection(segment.ray);
	if (intersection.shapeId >= 0)
	{
		// TODO: compute normal and uv
		glm::vec3 normal;
		glm::vec2 uv;

		ShadeableIntersection shadeable;
		shadeable.t = intersection.t;
		shadeable.position = segment.ray * intersection.t;
		glm::ivec4 idx = scene.dev_triangles[intersection.shapeId];

		glm::vec3 v[3]{ scene.dev_vertices[idx.x], scene.dev_vertices[idx.y], scene.dev_vertices[idx.z] };
		shadeable.normal = glm::normalize(glm::cross(v[1] - v[0], v[2] - v[0]));
		shadeable.uv = uv;
		shadeable.materialId = intersection.materialId; // TODO: change to materialId

		shadeable_intersection = shadeable;
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].throughput *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.normal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].throughput *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].throughput *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].throughput = glm::vec3(0.0f);
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(float u, int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= nPaths) return;

	PathSegment segment = iterationPaths[index];

	glm::vec3 pre_color = image[segment.pixelIndex];
	glm::vec3 new_color = glm::mix(pre_color, segment.radiance, u);

	image[segment.pixelIndex] = new_color;
}

// Naive BSDF sample only
__global__ void KernelNaiveGI(int iteration, int num_paths, 
							ShadeableIntersection* shadeableIntersections,
							PathSegment* pathSegments,
							Material* materials)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;

	ShadeableIntersection intersection = shadeableIntersections[idx];
	
	if (intersection.materialId >= 0)
	{
		pathSegments[idx].radiance = intersection.normal * 0.5f + 0.5f;
		//PathSegment segment = pathSegments[idx];
		//
		//Material material = materials[intersection.materialId];
		//glm::vec3 materialColor = material.color;
		//
		//if (material.emittance > 0.f) 
		//{
		//	glm::vec3 final_throughput = segment.throughput * material.emittance;
		//	pathSegments[idx].radiance = final_throughput;
		//	pathSegments[idx].Terminate();
		//}
		//else
		//{		
		//	glm::vec3 wo = WorldToLocal(intersection.normal) * -segment.ray.direction;
		//	if (wo.z < 0.f)
		//	{
		//		pathSegments[idx].Terminate();
		//		return;
		//	}
		//
		//	thrust::default_random_engine rng = makeSeededRandomEngine(iteration, idx, 0);
		//	thrust::uniform_real_distribution<float> u01(0.f, 1.f);
		//
		//	// naive diffuse surface
		//	glm::vec2 xi(u01(rng), u01(rng));
		//	glm::vec3 wi = SquareToHemisphereCosine(xi);
		//	glm::vec3 wiW = glm::normalize(LocalToWorld(intersection.normal) * wi);
		//
		//	float pdf = SquareToHemisphereCosinePDF(wi);
		//
		//	// generate new ray
		//	pathSegments[idx].ray = Ray::SpawnRay(intersection.position, wiW);
		//	pathSegments[idx].throughput *= materialColor * InvPi * wi.z / pdf;
		//}
	}
	else
	{
		pathSegments[idx].Terminate();
	}
}

CPU_ONLY void CudaPathTracer::Init(Scene* scene)
{
	m_Iteration = 0;
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_hdr_img, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_hdr_img, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_terminated_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need

	thrust_dev_paths = thrust::device_ptr<PathSegment>(dev_paths);
	thrust_dev_terminated_paths = thrust::device_ptr<PathSegment>(dev_terminated_paths);

	checkCUDAError("pathtraceInit");
}

CPU_ONLY void CudaPathTracer::GetImage(uchar4* host_image)
{
	const Camera& cam = hst_scene->state.camera;
	// Retrieve image from GPU
	hipMemcpy(host_image, dev_img, cam.resolution.x * cam.resolution.y * sizeof(uchar4), hipMemcpyDeviceToHost);
}

CPU_ONLY void CudaPathTracer::RegisterPBO(unsigned int pbo)
{
	hipGraphicsGLRegisterBuffer(&cuda_pbo_dest_resource, pbo, cudaGraphicsMapFlagsNone);
	size_t byte_count = resolution.x * resolution.y * 4 * sizeof(uchar4);
	hipGraphicsMapResources(1, &cuda_pbo_dest_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&dev_img, &byte_count, cuda_pbo_dest_resource);
	checkCUDAError("Get PBO pointer Error");
}

CPU_ONLY void CudaPathTracer::Render(GPUScene& scene)
{
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// TODO: might change to dynamic block size
	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, m_Iteration, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (depth < 3 && num_paths > 0) 
	{
		depth++;

		// clean shading chunks
		//hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_paths, dev_intersections, scene);
		checkCUDAError("Intersection Error");
		hipDeviceSynchronize();

		KernelNaiveGI<<<numblocksPathSegmentTracing, blockSize1d >>>(m_Iteration, num_paths,
																		dev_intersections, dev_paths, dev_materials);
		checkCUDAError("NaiveGI Error");
		hipDeviceSynchronize();

		if (guiData != nullptr)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;

	float u = 1.f / static_cast<float>(m_Iteration + 1); // used for interpolation between last frame and this frame

	finalGather << <numBlocksPixels, blockSize1d >> > (u, num_paths, dev_hdr_img, dev_paths);
	checkCUDAError("Final Gather failed");
	hipDeviceSynchronize();
	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (dev_img, cam.resolution, dev_hdr_img);

	checkCUDAError("pathtrace");
	++m_Iteration;
}