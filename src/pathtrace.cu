#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/partition.h>
#include <thrust/device_vector.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

struct continue_ray
{
	__host__ __device__
		bool operator()(const PathSegment& p)
	{
		return p.remainingBounces != 0;
	}
};

struct sort_on_material_id
{
	__host__ __device__
		bool operator()(const ShadeableIntersection& s1, const ShadeableIntersection& s2)
	{
		return s1.materialId < s2.materialId;
	}
};

__host__ __device__ glm::vec2 sampleDiscConcentric(const glm::vec2& in) {
	glm::vec2 offset = 2.f * in - glm::vec2(1.f, 1.f);
	if (offset.x == 0 && offset.y == 0) {
		return glm::vec2(0.f, 0.f);
	}
	float r, theta;
	if (std::abs(offset.x) > std::abs(offset.y)) {
		r = offset.x;
		theta = PI_OVER_FOUR * (offset.y / offset.x);
	}
	else {
		r = offset.y;
		theta = PI_OVER_TWO - PI_OVER_FOUR * (offset.x / offset.y);
	}
	return r * glm::vec2(cos(theta), sin(theta));
}


__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
#if FIRST_BOUNCE_CACHED
static ShadeableIntersection* dev_intersections_cached = NULL;
#endif
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

#if FIRST_BOUNCE_CACHED
	hipMalloc(&dev_intersections_cached, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections_cached, 0, pixelcount * sizeof(ShadeableIntersection));
#endif

	// TODO: initialize any extra device memeory you need

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
#if FIRST_BOUNCE_CACHED
	hipFree(dev_intersections_cached);
#endif
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(0.0f, 0.0f, 0.0f);
		segment.accumCol = glm::vec3(1.0f, 1.0f, 1.0f);

		float dX = 0, dY = 0;

#if ANTI_ALIASING || DEPTH_OF_FIELD
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
		thrust::uniform_real_distribution<float> uX(0, 1);
		thrust::uniform_real_distribution<float> uY(0, 1);
#endif

#if ANTI_ALIASING			
		dX = uX(rng); dY = uY(rng);
#endif

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)(x + dX) - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)(y + dY) - (float)cam.resolution.y * 0.5f)
		);

#if DEPTH_OF_FIELD
		//Entirely referenced PBRT : https://pbr-book.org/3ed-2018/Camera_Models/Projective_Camera_Models#TheThinLensModelandDepthofField

		glm::vec2 pointOnLens = cam.lensRadius * sampleDiscConcentric(glm::vec2(uX(rng), uY(rng)));

		//How I got this t-value?
		//Eq 1: cam.pos + t * ray.dir = pointOnFilm
		//Eq 2: (pointOnFilm - focalLength * view).(z-axis) = 0. This is assuming focal pplane to be an XY plane so a dot of its normal with a vector on it will be zero
		//Solve for t. PBRT equation (presumably) assumes view vector to align with the z axis and the lens to be at the origin
		float t = cam.focalLength * glm::dot(cam.view, cam.view)/glm::dot(segment.ray.direction, cam.view);// (cam.focalLength * cam.view.z - cam.position.z) / segment.ray.direction.z;
		glm::vec3 pointOnFilm = getPointOnRay(segment.ray, t);

		segment.ray.origin += glm::vec3(pointOnLens.x, pointOnLens.y, 0);
		segment.ray.direction = glm::normalize(pointOnFilm - segment.ray.origin);
#endif

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
			pathSegment.remainingBounces = 0;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void kernShadeAllMaterials(
	int iter
	, int depth
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces == 0)
			return;
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) {
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color = (materialColor * material.emittance) * pathSegments[idx].accumCol;
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				scatterRay(pathSegments[idx],
					getPointOnRay(pathSegments[idx].ray, intersection.t),
					intersection.surfaceNormal,
					material,
					rng);				
			}
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, num_paths * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

#if FIRST_BOUNCE_CACHED
		if (iter == 1 && depth == 0) {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				);
			checkCUDAError("trace first cached bounce");
			hipDeviceSynchronize();
			hipMemcpy(dev_intersections_cached, dev_intersections, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else if (depth == 0) {
			hipMemcpy(dev_intersections, dev_intersections_cached, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else
#endif
		{
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}
		depth++;		

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
	  // evaluating the BSDF.
	  // Start off with just a big kernel that handles all the different
	  // materials you have in the scenefile.
	  // TODO: compare between directly shading the path segments and shading
	  // path segments that have been reshuffled to be contiguous in memory.

#if MATERIAL_SORT
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, sort_on_material_id());
#endif
		kernShadeAllMaterials << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			depth,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
			);

		hipDeviceSynchronize();

#if STREAM_COMPACTION
		thrust::device_ptr<PathSegment> thrust_dev_paths(dev_paths);
		thrust::device_ptr<PathSegment> end = thrust::stable_partition(thrust::device, thrust_dev_paths, thrust_dev_paths + num_paths, continue_ray());
		num_paths = thrust::distance(thrust_dev_paths, end);
#endif

		iterationComplete = (depth == traceDepth) || (num_paths == 0);

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
