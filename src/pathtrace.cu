#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1


#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

__device__ inline bool util_math_is_nan(const glm::vec3& v)
{
	return (v.x != v.x) || (v.y != v.y) || (v.z != v.z);
}



//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::vec3 color;
#if TONEMAPPING
		color = pix / (float)iter;
		color = util_postprocess_gamma(util_postprocess_ACESFilm(color));
		color = color * 255.0f;
#else
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);
#endif
		if (util_math_is_nan(pix))
		{
			pbo[index].x = 255;
			pbo[index].y = 192;
			pbo[index].z = 203;
		}
		else
		{
			// Each thread writes one pixel location in the texture (textel)
			pbo[index].x = color.x;
			pbo[index].y = color.y;
			pbo[index].z = color.z;
		}
		pbo[index].w = 0;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Object* dev_objs = NULL;
static Material* dev_materials = NULL;
static BVHGPUNode* dev_bvhArray = NULL;
static MTBVHGPUNode* dev_mtbvhArray = NULL;
static Primitive* dev_primitives = NULL;
static glm::ivec3* dev_triangles = NULL;
static glm::vec3* dev_vertices = NULL;
static glm::vec2* dev_uvs = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec3* dev_tangents = NULL;
static float* dev_fsigns = NULL;
static PathSegment* dev_paths1 = NULL;
static PathSegment* dev_paths2 = NULL;
static ShadeableIntersection* dev_intersections1 = NULL;
static ShadeableIntersection* dev_intersections2 = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths1, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_paths2, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_objs, scene->objects.size() * sizeof(Object));
	hipMemcpy(dev_objs, scene->objects.data(), scene->objects.size() * sizeof(Object), hipMemcpyHostToDevice);

	if (scene->triangles.size())
	{
		hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(glm::ivec3));
		hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(glm::ivec3), hipMemcpyHostToDevice);

		hipMalloc(&dev_vertices, scene->verticies.size() * sizeof(glm::vec3));
		hipMemcpy(dev_vertices, scene->verticies.data(), scene->verticies.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

		hipMalloc(&dev_uvs, scene->uvs.size() * sizeof(glm::vec2));
		hipMemcpy(dev_uvs, scene->uvs.data(), scene->uvs.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
		if (scene->normals.size())
		{
			hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
			hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
		}
		if (scene->tangents.size())
		{
			hipMalloc(&dev_tangents, scene->tangents.size() * sizeof(glm::vec3));
			hipMemcpy(dev_tangents, scene->tangents.data(), scene->tangents.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
		}
		if (scene->fSigns.size())
		{
			hipMalloc(&dev_fsigns, scene->fSigns.size() * sizeof(float));
			hipMemcpy(dev_fsigns, scene->fSigns.data(), scene->fSigns.size() * sizeof(float), hipMemcpyHostToDevice);
		}
	}
#if MTBVH
	hipMalloc(&dev_mtbvhArray, scene->MTBVHArray.size() * sizeof(MTBVHGPUNode));
	hipMemcpy(dev_mtbvhArray, scene->MTBVHArray.data(), scene->MTBVHArray.size() * sizeof(MTBVHGPUNode), hipMemcpyHostToDevice);
#else
	hipMalloc(&dev_bvhArray, scene->bvhArray.size() * sizeof(BVHGPUNode));
	hipMemcpy(dev_bvhArray, scene->bvhArray.data(), scene->bvhArray.size() * sizeof(BVHGPUNode), hipMemcpyHostToDevice);
#endif

	hipMalloc(&dev_primitives, scene->primitives.size() * sizeof(Primitive));
	hipMemcpy(dev_primitives, scene->primitives.data(), scene->primitives.size() * sizeof(Primitive), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections1, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections1, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_intersections2, pixelcount * sizeof(ShadeableIntersection));
	// TODO: initialize any extra device memeory you need

	checkCUDAError("pathtraceInit");
}

void pathtraceFree(Scene* scene) {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths1);
	hipFree(dev_paths2);
	hipFree(dev_objs);
	if (scene->triangles.size())
	{
		hipFree(dev_triangles);
		hipFree(dev_vertices);
		hipFree(dev_uvs);
		if (scene->normals.size())
		{
			hipFree(dev_normals);
		}
		if (scene->tangents.size())
		{
			hipFree(dev_tangents);
		}
		if (scene->fSigns.size())
		{
			hipFree(dev_fsigns);
		}
	}
	hipFree(dev_primitives);
#if MTBVH
	hipFree(dev_mtbvhArray);
#else
	hipFree(dev_bvhArray);
#endif
	hipFree(dev_materials);
	hipFree(dev_intersections1);
	hipFree(dev_intersections2);
	// TODO: clean up any extra device memory you created

	checkCUDAError("pathtraceFree");
}

__device__ inline glm::vec2 util_concentric_sample_disk(glm::vec2 rand)
{
	rand = 2.0f * rand - 1.0f;
	if (rand.x == 0 && rand.y == 0)
	{
		return glm::vec2(0);
	}
	const float pi_4 = PI / 4, pi_2 = PI / 2;
	bool x_g_y = abs(rand.x) > abs(rand.y);
	float theta = x_g_y ? pi_4 * rand.y / rand.x : pi_2 - pi_4 * rand.x / rand.y;
	float r = x_g_y ? rand.x : rand.y;
	return glm::vec2(cos(theta), sin(theta)) * r;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	thrust::default_random_engine rng = makeSeededRandomEngine(x, y, iter);
	thrust::uniform_real_distribution<float> u01(0, 1);

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
#if STOCHASTIC_SAMPLING
		// TODO: implement antialiasing by jittering the ray
		glm::vec2 jitter = glm::vec2(0.5f * (u01(rng) * 2.0f - 1.0f), 0.5f * (u01(rng) * 2.0f - 1.0f));
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + jitter[0])
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + jitter[1])
		);
#if DOF_ENABLED
		float lensR = cam.lensRadius;
		glm::vec3 perpDir = glm::cross(cam.right, cam.up);
		perpDir = glm::normalize(perpDir);
		float focalLen = cam.focalLength;
		float tFocus = focalLen / glm::abs(glm::dot(segment.ray.direction, perpDir));
		glm::vec2 offset = lensR * util_concentric_sample_disk(glm::vec2(u01(rng), u01(rng)));
		glm::vec3 newOri = offset.x * cam.right + offset.y * cam.up + cam.position;
		glm::vec3 pFocus = segment.ray.direction * tFocus + segment.ray.origin;
		segment.ray.direction = glm::normalize(pFocus - newOri);
		segment.ray.origin = newOri;
#endif

#else
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#endif
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void compute_intersection(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Object* geoms
	, int objs_size
	, glm::ivec3* modelTriangles
	, glm::vec3* modelVertices
	, const glm::vec2* modelUVs
	, const glm::vec3* modelNormals
	, hipTextureObject_t skyboxTex
	, ShadeableIntersection* intersections
	, int* rayValid
	, glm::vec3* image
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment& pathSegment = pathSegments[path_index];
		float t = -1.0;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_material_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < objs_size; i++)
		{
			Object& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal);
			}
			else if (geom.type == SPHERE)
			{
				t = util_geometry_ray_sphere_intersection(geom, pathSegment.ray, tmp_intersect, tmp_normal);
			}
			else if (geom.type == TRIANGLE_MESH)
			{
				glm::vec3 baryCoord;
				for (int i = geom.triangleStart; i != geom.triangleEnd; i++)
				{
					const glm::ivec3& tri = modelTriangles[i];
					const glm::vec3& v0 = modelVertices[tri[0]];
					const glm::vec3& v1 = modelVertices[tri[1]];
					const glm::vec3& v2 = modelVertices[tri[2]];
					t = triangleIntersectionTest(geom.Transform, v0, v1, v2, pathSegment.ray, tmp_intersect, tmp_normal, baryCoord);
					if (t > 0.0f && t_min > t)
					{
						t_min = t;
						hit_material_index = geom.materialid;
						intersect_point = tmp_intersect;
						normal = tmp_normal;
					}
				}
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_material_index = geom.materialid;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (t_min == FLT_MAX)//hits nothing
		{
			rayValid[path_index] = 0;
			if (skyboxTex)
			{
				glm::vec2 uv = util_sample_spherical_map(glm::normalize(pathSegment.ray.direction));
				float4 skyColorRGBA = tex2D<float4>(skyboxTex, uv.x, uv.y);
				glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
				image[pathSegment.pixelIndex] += pathSegment.color * skyColor;
			}
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = hit_material_index;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].worldPos = intersect_point;
			rayValid[path_index] = 1;
		}
	}
}

__device__ inline int util_bvh_get_sibling(const BVHGPUNode* bvhArray, int curr)
{
	int parent = bvhArray[curr].parent;
	if (parent == -1) return -1;
	return bvhArray[parent].left == curr ? bvhArray[parent].right : bvhArray[parent].left;
}

__device__ inline int util_bvh_get_near_child(const BVHGPUNode* bvhArray, int curr, const glm::vec3& rayDir)
{
	return rayDir[bvhArray[curr].axis] > 0.0 ? bvhArray[curr].left : bvhArray[curr].right;
}

__device__ inline bool util_bvh_is_leaf(const BVHGPUNode* bvhArray, int curr)
{
	return bvhArray[curr].left == -1 && bvhArray[curr].right == -1;
}

__device__ inline bool util_bvh_leaf_intersect(const Primitive* primitives, 
	int primsStart, 
	int primsEnd, 
	const Object* objects, 
	const glm::ivec3* modelTriangles, 
	const  glm::vec3* modelVertices, 
	const  glm::vec2* modelUVs,
	const glm::vec3* modelNormals, 
	const glm::vec3* modelTangents,
	const float* modelFsigns,
	const Ray& ray, 
	ShadeableIntersection* intersection
)
{
	glm::vec3 tmp_intersect, tmp_normal, tmp_baryCoord, tmp_tangent;
	float tmp_fsign;
	glm::vec2	tmp_uv;
	bool intersected = false;
	float t = -1.0;
	for (int i = primsStart; i != primsEnd; i++)
	{
		const Primitive& prim = primitives[i];
		int objID = prim.objID;
		const Object& obj = objects[objID];
		
		if (obj.type == TRIANGLE_MESH)
		{
			const glm::ivec3& tri = modelTriangles[obj.triangleStart + prim.offset];
			const glm::vec3& v0 = modelVertices[tri[0]];
			const glm::vec3& v1 = modelVertices[tri[1]];
			const glm::vec3& v2 = modelVertices[tri[2]];
			t = triangleIntersectionTest(obj.Transform, v0, v1, v2, ray, tmp_intersect, tmp_normal, tmp_baryCoord);
			if (modelNormals && modelUVs)
			{
				const glm::vec3& n0 = modelNormals[tri[0]];
				const glm::vec3& n1 = modelNormals[tri[1]];
				const glm::vec3& n2 = modelNormals[tri[2]];
				tmp_normal = n0 * tmp_baryCoord[0] + n1 * tmp_baryCoord[1] + n2 * tmp_baryCoord[2];
				tmp_normal = glm::vec3(obj.Transform.invTranspose * glm::vec4(tmp_normal, 0.0));//TODO: precompute transformation
				const glm::vec2& uv0 = modelUVs[tri[0]];
				const glm::vec2& uv1 = modelUVs[tri[1]];
				const glm::vec2& uv2 = modelUVs[tri[2]];
				tmp_uv = uv0 * tmp_baryCoord[0] + uv1 * tmp_baryCoord[1] + uv2 * tmp_baryCoord[2];
				const glm::vec3& t0 = modelTangents[tri[0]];
				const glm::vec3& t1 = modelTangents[tri[1]];
				const glm::vec3& t2 = modelTangents[tri[2]];
				tmp_tangent = t0 * tmp_baryCoord[0] + t1 * tmp_baryCoord[1] + t2 * tmp_baryCoord[2];
				tmp_tangent = glm::vec3(obj.Transform.invTranspose * glm::vec4(tmp_tangent, 0.0));
			}
		}
		else if (obj.type == CUBE)
		{
			t = boxIntersectionTest(obj, ray, tmp_intersect, tmp_normal);
		}
		else if (obj.type == SPHERE)
		{
			t = util_geometry_ray_sphere_intersection(obj, ray, tmp_intersect, tmp_normal);
		}
		
		if (t > 0.0 && t < intersection->t)
		{
			intersection->t = t;
			intersection->materialId = obj.materialid;
			intersection->worldPos = tmp_intersect;
			intersection->surfaceNormal = tmp_normal;
			intersection->surfaceTangent = tmp_tangent;
			intersection->fsign = tmp_fsign;
			intersection->uv = tmp_uv;
			intersected = true;
		}
		
	}
	return intersected;
}

enum bvh_traverse_state {
	fromChild,fromParent,fromSibling
};

__global__ void compute_intersection_bvh_stackless(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Material* materials
	, const Object* objects
	, int objs_size
	, const glm::ivec3* modelTriangles
	, const glm::vec3* modelVertices
	, const glm::vec2* modelUVs
	, const glm::vec3* modelNormals
	, const glm::vec3* modelTangents
	, const float* modelFsigns
	, hipTextureObject_t skyboxTex
	, const Primitive* primitives
	, const BVHGPUNode* bvhArray
	, int bvhArraySize
	, ShadeableIntersection* intersections
	, int* rayValid
	, glm::vec3* image
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index >= num_paths) return;
	PathSegment& pathSegment = pathSegments[path_index];
	glm::vec3 rayDir = pathSegment.ray.direction;
	glm::vec3 rayOri = pathSegment.ray.origin;
	int curr = util_bvh_get_near_child(bvhArray, 0, rayOri);
	bvh_traverse_state state = fromParent;
	ShadeableIntersection tmpIntersection;
	tmpIntersection.t = 1e37f;
	bool intersected = false;
	while (curr >= 0 && curr < bvhArraySize)
	{
		if (state == fromChild)
		{
			if (curr == 0) break;
			int parent = bvhArray[curr].parent;
			if (curr == util_bvh_get_near_child(bvhArray, parent, rayOri))
			{
				curr = util_bvh_get_sibling(bvhArray, curr);
				state = fromSibling;
			}
			else
			{
				curr = parent;
				state = fromChild;
			}
		}
		else if (state == fromSibling)
		{
			bool outside = true;
			float boxt = boundingBoxIntersectionTest(bvhArray[curr].bbox, pathSegment.ray, outside);
			if (!outside) boxt = EPSILON;
			if (!(boxt > 0 && boxt < tmpIntersection.t))
			{
				curr = bvhArray[curr].parent;
				state = fromChild;
			}
			else if (util_bvh_is_leaf(bvhArray, curr))
			{
				int start = bvhArray[curr].startPrim, end = bvhArray[curr].endPrim;
				if (util_bvh_leaf_intersect(primitives, start, end, objects, modelTriangles, modelVertices, modelUVs, modelNormals, modelTangents, modelFsigns, pathSegment.ray, &tmpIntersection))
				{
					intersected = true;
				}
				curr = bvhArray[curr].parent;
				state = fromChild;
			}
			else
			{
				curr = util_bvh_get_near_child(bvhArray, curr, rayOri);
				state = fromParent;
			}
		}
		else// from parent
		{
			bool outside = true;
			float boxt = boundingBoxIntersectionTest(bvhArray[curr].bbox, pathSegment.ray, outside);
			if (!outside) boxt = EPSILON;
			if (!(boxt > 0 && boxt < tmpIntersection.t))
			{
				curr = util_bvh_get_sibling(bvhArray, curr);
				state = fromSibling;
			}
			else if (util_bvh_is_leaf(bvhArray, curr))
			{
				int start = bvhArray[curr].startPrim, end = bvhArray[curr].endPrim;
				if (util_bvh_leaf_intersect(primitives, start, end, objects, modelTriangles, modelVertices, modelUVs, modelNormals, modelTangents, modelFsigns, pathSegment.ray, &tmpIntersection))
				{
					intersected = true;
				}
				curr = util_bvh_get_sibling(bvhArray, curr);
				state = fromSibling;
			}
			else
			{
				curr = util_bvh_get_near_child(bvhArray, curr, pathSegment.ray.origin);
				state = fromParent;
			}
		}
	}
	rayValid[path_index] = intersected;
	if (intersected)
	{
		intersections[path_index] = tmpIntersection;
		intersections[path_index].type = materials[tmpIntersection.materialId].type;
	}
	else if(skyboxTex)
	{
		glm::vec2 uv = util_sample_spherical_map(glm::normalize(rayDir));
		float4 skyColorRGBA = tex2D<float4>(skyboxTex, uv.x, uv.y);
		glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
		image[pathSegment.pixelIndex] += pathSegment.color * skyColor * BACKGROUND_COLOR_MULT;
	}
}



__global__ void compute_intersection_bvh_stackless_mtbvh(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Material* materials
	, const Object* objects
	, int objs_size
	, const glm::ivec3* modelTriangles
	, const glm::vec3* modelVertices
	, const glm::vec2* modelUVs
	, const glm::vec3* modelNormals
	, const glm::vec3* modelTangents
	, const float* modelFsigns
	, hipTextureObject_t skyboxTex
	, const Primitive* primitives
	, const MTBVHGPUNode* bvhArray
	, int bvhArraySize
	, ShadeableIntersection* intersections
	, int* rayValid
	, glm::vec3* image
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index >= num_paths) return;
	PathSegment& pathSegment = pathSegments[path_index];
	Ray& ray = pathSegment.ray;
	glm::vec3 rayDir = pathSegment.ray.direction;
	glm::vec3 rayOri = pathSegment.ray.origin;
	float x = fabs(rayDir.x), y = fabs(rayDir.y), z = fabs(rayDir.z);
	int axis = x > y && x > z ? 0 : (y > z ? 1 : 2);
	int sgn = rayDir[axis] > 0 ? 0 : 1;
	int d = (axis << 1) + sgn;
	const MTBVHGPUNode* currArray = bvhArray + d * bvhArraySize;
	int curr = 0;
	ShadeableIntersection tmpIntersection;
	tmpIntersection.t = 1e37f;
	bool intersected = false;
	while (curr >= 0 && curr < bvhArraySize)
	{
		bool outside = true;
		float boxt = boundingBoxIntersectionTest(currArray[curr].bbox, ray, outside);
		if (!outside) boxt = EPSILON;
		if (boxt > 0 && boxt < tmpIntersection.t)
		{
			if (currArray[curr].startPrim != -1)//leaf node
			{
				int start = currArray[curr].startPrim, end = currArray[curr].endPrim;
				bool intersect = util_bvh_leaf_intersect(primitives, start, end, objects, modelTriangles, modelVertices, modelUVs, modelNormals, modelTangents, modelFsigns, ray, &tmpIntersection);
				intersected = intersected || intersect;
			}
			curr = currArray[curr].hitLink;
		}
		else
		{
			curr = currArray[curr].missLink;
		}
	}
	
	rayValid[path_index] = intersected;
	if (intersected)
	{
		intersections[path_index] = tmpIntersection;
		intersections[path_index].type = materials[tmpIntersection.materialId].type;
		pathSegment.remainingBounces--;
	}
	else if (skyboxTex)
	{
		glm::vec2 uv = util_sample_spherical_map(glm::normalize(rayDir));
		float4 skyColorRGBA = tex2D<float4>(skyboxTex, uv.x, uv.y);
		glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
		image[pathSegment.pixelIndex] += depth > 0 ? pathSegment.color * skyColor * BACKGROUND_COLOR_MULT : pathSegment.color * skyColor;
	}
}


__global__ void scatter_on_intersection(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
	, int* rayValid
	, glm::vec3* image
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;
	ShadeableIntersection intersection = shadeableIntersections[idx];
	// Set up the RNG
	// LOOK: this is how you use thrust's RNG! Please look at
	// makeSeededRandomEngine as well.
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
	thrust::uniform_real_distribution<float> u01(0, 1);

	Material material = materials[intersection.materialId];
	glm::vec3 materialColor = material.color;
#if VIS_NORMAL
	image[pathSegments[idx].pixelIndex] += (glm::normalize(intersection.surfaceNormal));
	rayValid[idx] = 0;
	return;
#endif

	// If the material indicates that the object was a light, "light" the ray
	if (material.type == MaterialType::emitting) {
		pathSegments[idx].color *= (materialColor * material.emittance);
		rayValid[idx] = 0;
		if (!util_math_is_nan(pathSegments[idx].color))
			image[pathSegments[idx].pixelIndex] += pathSegments[idx].color;
	}
	else {
		glm::vec3& woInWorld = pathSegments[idx].ray.direction;
		glm::vec3 nMap = glm::vec3(0, 0, 1);
		if (material.normalMap != 0)
		{
			float4 nMapCol = tex2D<float4>(material.normalMap, intersection.uv.x, 1.0 - intersection.uv.y);
			nMap.x = nMapCol.x;
			nMap.y = nMapCol.y;
			nMap.z = nMapCol.z;
			nMap=glm::pow(nMap, glm::vec3(1 / 2.2f));
			nMap = nMap * 2.0f - 1.0f;
			nMap = glm::normalize(nMap);
		}
		
		glm::vec3 N = glm::normalize(intersection.surfaceNormal);
		glm::vec3 B, T;
		if (material.normalMap != 0)
		{
			T = intersection.surfaceTangent;
			T = glm::normalize(T - N * glm::dot(N, T));
			B = glm::cross(N, T);
			N = glm::normalize(T * nMap.x + B * nMap.y + N * nMap.z);
		}
		else
		{
			util_math_get_TBN_pixar(N, &T, &B);
		}
		glm::mat3 TBN(T, B, N);
		glm::vec3 wo = glm::transpose(TBN) * (-woInWorld);
		wo = glm::normalize(wo);
		float pdf = 0;
		glm::vec3 wi, bxdf;
		glm::vec3 random = glm::vec3(u01(rng), u01(rng), u01(rng));
		if (material.type == MaterialType::metallicWorkflow)
		{
			float4 color = { 0,0,0,1 };
			float roughness = material.roughness, metallic = material.metallic;
			if (material.baseColorMap != 0)
			{
				color = tex2D<float4>(material.baseColorMap, intersection.uv.x, 1.0 - intersection.uv.y);
				materialColor.x = color.x;
				materialColor.y = color.y;
				materialColor.z = color.z;
			}
			if (material.metallicRoughnessMap != 0)
			{
				color = tex2D<float4>(material.metallicRoughnessMap, intersection.uv.x, 1.0 - intersection.uv.y);
				roughness = color.y;
				metallic = color.z;
			}
			
			bxdf = bxdf_metallic_workflow_sample_f(wo, &wi, random, &pdf, materialColor, metallic, roughness);
		}
		else if (material.type == MaterialType::frenselSpecular)
		{
			glm::vec2 iors = glm::dot(woInWorld, N) < 0 ? glm::vec2(1.0, material.indexOfRefraction) : glm::vec2(material.indexOfRefraction, 1.0);
			bxdf = bxdf_frensel_specular_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, materialColor, iors);
		}
		else if (material.type == MaterialType::microfacet)
		{
			bxdf = bxdf_microfacet_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, material.roughness);
		}
		else//diffuse
		{
			float4 color = { 0,0,0,1 };
			if (material.baseColorMap != 0)
			{
				color = tex2D<float4>(material.baseColorMap, intersection.uv.x, intersection.uv.y);
				materialColor.x = color.x;
				materialColor.y = color.y;
				materialColor.z = color.z;
			}
			if (color.w <= ALPHA_CUTOFF)
			{
				bxdf = pathSegments[idx].remainingBounces == 0 ? glm::vec3(0, 0, 0) : glm::vec3(1, 1, 1);
				wi = -wo;
				pdf = util_math_tangent_space_abscos(wi);
			}
			else
			{
				bxdf = bxdf_diffuse_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor);
			}
			
		}
		if (pdf > 0)
		{
			pathSegments[idx].color *= bxdf * util_math_tangent_space_abscos(wi) / pdf;
			glm::vec3 newDir = glm::normalize(TBN * wi);
			glm::vec3 offset = glm::dot(newDir, N) < 0 ? -N : N;
			float offsetMult = material.type != MaterialType::frenselSpecular ? SCATTER_ORIGIN_OFFSETMULT : SCATTER_ORIGIN_OFFSETMULT * 100.0f;
			pathSegments[idx].ray.origin = intersection.worldPos + offset * offsetMult;
			pathSegments[idx].ray.direction = newDir;
			rayValid[idx] = 1;
		}
		else
		{
			rayValid[idx] = 0;
		}

	}
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		if (!util_math_is_nan(iterationPath.color))
			image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

struct mat_comp {
	__host__ __device__ bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
		return a.type < b.type;
	}
};

int compact_rays(int* rayValid,int* rayIndex,int numRays, bool sortByMat=false)
{
	thrust::device_ptr<PathSegment> dev_thrust_paths1(dev_paths1), dev_thrust_paths2(dev_paths2);
	thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections1(dev_intersections1), dev_thrust_intersections2(dev_intersections2);
	thrust::device_ptr<int> dev_thrust_rayValid(rayValid), dev_thrust_rayIndex(rayIndex);
	thrust::exclusive_scan(dev_thrust_rayValid, dev_thrust_rayValid + numRays, dev_thrust_rayIndex);
	int nextNumRays, tmp;
	hipMemcpy(&tmp, rayIndex + numRays - 1, sizeof(int), hipMemcpyDeviceToHost);
	nextNumRays = tmp;
	hipMemcpy(&tmp, rayValid + numRays - 1, sizeof(int), hipMemcpyDeviceToHost);
	nextNumRays += tmp;
	thrust::scatter_if(dev_thrust_paths1, dev_thrust_paths1 + numRays, dev_thrust_rayIndex, dev_thrust_rayValid, dev_thrust_paths2);
	thrust::scatter_if(dev_thrust_intersections1, dev_thrust_intersections1 + numRays, dev_thrust_rayIndex, dev_thrust_rayValid, dev_thrust_intersections2);
	if (sortByMat)
	{
		mat_comp cmp;
		thrust::sort_by_key(dev_thrust_intersections2, dev_thrust_intersections2 + nextNumRays, dev_thrust_paths2, cmp);
	}
	std::swap(dev_paths1, dev_paths2);
	std::swap(dev_intersections1, dev_intersections2);
	return nextNumRays;
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, MAX_ITER, dev_paths1);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths1 + pixelcount;
	int num_paths = dev_path_end - dev_paths1;
	int* rayValid, * rayIndex;
	
	int numRays = num_paths;
	hipMalloc((void**)&rayValid, sizeof(int) * pixelcount);
	hipMalloc((void**)&rayIndex, sizeof(int) * pixelcount);
	
	hipDeviceSynchronize();
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (numRays && depth < MAX_ITER) {

		// clean shading chunks
		hipMemset(dev_intersections1, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (numRays + blockSize1d - 1) / blockSize1d;
#if USE_BVH
#if MTBVH
		compute_intersection_bvh_stackless_mtbvh << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, numRays
			, dev_paths1
			, dev_materials
			, dev_objs
			, hst_scene->objects.size()
			, dev_triangles
			, dev_vertices
			, dev_uvs
			, dev_normals
			, dev_tangents
			, dev_fsigns
			, hst_scene->skyboxTextureObj
			, dev_primitives
			, dev_mtbvhArray
			, hst_scene->bvhTreeSize
			, dev_intersections1
			, rayValid
			, dev_image
			);
#else
		compute_intersection_bvh_stackless << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, numRays
			, dev_paths1
			, dev_materials
			, dev_objs
			, hst_scene->objects.size()
			, dev_triangles
			, dev_vertices
			, dev_uvs
			, dev_normals
			, dev_tangents
			, hst_scene->skyboxTextureObj
			, dev_primitives
			, dev_bvhArray
			, hst_scene->bvhArray.size()
			, dev_intersections1
			, rayValid
			, dev_image
			);
#endif
#else
		compute_intersection << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, numRays
			, dev_paths1
			, dev_objs
			, hst_scene->objects.size()
			, dev_triangles
			, dev_vertices
			, dev_uvs
			, dev_normals
			, hst_scene->skyboxTextureObj
			, dev_intersections1
			, rayValid
			, dev_image
			);
#endif
		hipDeviceSynchronize();
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

#if SORT_BY_MATERIAL_TYPE
		numRays = compact_rays(rayValid, rayIndex, numRays, true);
#else
		numRays = compact_rays(rayValid, rayIndex, numRays);
#endif

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.
		if (!numRays) break;
		dim3 numblocksLightScatter = (numRays + blockSize1d - 1) / blockSize1d;
		scatter_on_intersection << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			numRays,
			dev_intersections1,
			dev_paths1,
			dev_materials,
			rayValid,
			dev_image
			);

		numRays = compact_rays(rayValid, rayIndex, numRays);

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	if (numRays)
	{
		// Assemble this iteration and apply it to the image
		dim3 numBlocksPixels = (numRays + blockSize1d - 1) / blockSize1d;
		finalGather << <numBlocksPixels, blockSize1d >> > (numRays, dev_image, dev_paths1);
	}

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	hipFree(rayValid);
	hipFree(rayIndex);

	checkCUDAError("pathtrace");
}
