#include "hip/hip_runtime.h"
#include "scene.h"
#include "sceneStructs.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

void Scene::traverseNode(const tinygltf::Model& model, int nodeIndex, const glm::mat4x4 & parentTransform)
{
    if (nodeIndex < 0 || nodeIndex >= model.nodes.size()) {
        return;
    }

    const auto& node = model.nodes[nodeIndex];
    
    glm::mat4 nodeTransform = parentTransform;
    applyNodeTransform(node, nodeTransform);

    if (node.mesh >= 0) {
        const auto& mesh = model.meshes[node.mesh];
        processMesh(model, mesh, nodeTransform);
    }

    // Recursively process child nodes
    for (int childIndex : node.children) {
        traverseNode(model, childIndex, nodeTransform);
    }
}

void Scene::processMesh(const tinygltf::Model& model, const tinygltf::Mesh& mesh, const glm::mat4x4 & transform)
{
    std::cout << "Loading mesh: " << mesh.name << std::endl;

    for (const auto& primitive : mesh.primitives) {
        int p_size = mesh.primitives.size();
        const auto& indicesAccessor = model.accessors[primitive.indices];
        const auto& positionsAccessor = model.accessors[primitive.attributes.at("POSITION")];
        const auto& normalsAccessor = model.accessors[primitive.attributes.at("NORMAL")];
        const auto& uvAccessor = model.accessors[primitive.attributes.at("TEXCOORD_0")];

        const auto& indicesView = model.bufferViews[indicesAccessor.bufferView];
        const auto& positionsView = model.bufferViews[positionsAccessor.bufferView];
        const auto& normalsView = model.bufferViews[normalsAccessor.bufferView];
        const auto& uvView = model.bufferViews[uvAccessor.bufferView];

        // TODO: Dynamic type array according to componentType
        const unsigned short* indexData = reinterpret_cast<const unsigned short*>(&model.buffers[indicesView.buffer].data[indicesAccessor.byteOffset + indicesView.byteOffset]);
        const float* positionData = reinterpret_cast<const float*>(&model.buffers[positionsView.buffer].data[positionsAccessor.byteOffset + positionsView.byteOffset]);
        const float* normalData = reinterpret_cast<const float*>(&model.buffers[normalsView.buffer].data[normalsAccessor.byteOffset + normalsView.byteOffset]);
        const float* uvData = reinterpret_cast<const float*>(&model.buffers[uvView.buffer].data[uvAccessor.byteOffset + uvView.byteOffset]);

        const size_t vertexStride = 3;
        const size_t normalStride = 3;
        const size_t uvStride = 3;

        const size_t numIndices = indicesAccessor.count;
        glm::mat4x4 normalTransform = glm::transpose(glm::inverse(transform));
        // Iterate through indices and create triangles
        //for (size_t i = 0; i < numIndices; i += 3) {
        for (size_t i = 0; i < numIndices; i += 3) {
            Triangle triangle;
            triangle.p1 = glm::vec3(positionData[indexData[i] * vertexStride], positionData[indexData[i] * vertexStride + 1], positionData[indexData[i] * vertexStride + 2]);
            triangle.p2 = glm::vec3(positionData[indexData[i + 1] * vertexStride], positionData[indexData[i + 1] * vertexStride + 1], positionData[indexData[i + 1] * vertexStride + 2]);
            triangle.p3 = glm::vec3(positionData[indexData[i + 2] * vertexStride], positionData[indexData[i + 2] * vertexStride + 1], positionData[indexData[i + 2] * vertexStride + 2]);
            
            triangle.p1 = glm::vec3(transform * glm::vec4(triangle.p1, 1.0f));
            triangle.p2 = glm::vec3(transform * glm::vec4(triangle.p2, 1.0f));
            triangle.p3 = glm::vec3(transform * glm::vec4(triangle.p3, 1.0f));

            triangle.n1 = glm::vec3(normalData[indexData[i] * normalStride], normalData[indexData[i] * normalStride + 1], normalData[indexData[i] * normalStride + 2]);
            triangle.n2 = glm::vec3(normalData[indexData[i + 1] * normalStride], normalData[indexData[i + 1] * normalStride + 1], normalData[indexData[i + 1] * normalStride + 2]);
            triangle.n3 = glm::vec3(normalData[indexData[i + 2] * normalStride], normalData[indexData[i + 2] * normalStride + 1], normalData[indexData[i + 2] * normalStride + 2]);

            triangle.n1 = glm::vec3(normalTransform * glm::vec4(triangle.n1, 0.0f));
            triangle.n1 = glm::vec3(normalTransform * glm::vec4(triangle.n1, 0.0f));
            triangle.n1 = glm::vec3(normalTransform * glm::vec4(triangle.n1, 0.0f));

            triangle.uv1 = glm::vec2(uvData[indexData[i] * uvStride], uvData[indexData[i] * uvStride + 1]);
            triangle.uv2 = glm::vec2(uvData[indexData[i + 1] * uvStride], uvData[indexData[i + 1] * uvStride + 1]);
            triangle.uv3 = glm::vec2(uvData[indexData[i + 2] * uvStride], uvData[indexData[i + 2] * uvStride + 1]);
            //auto index0 = indexData[i];
            //auto index1 = indexData[i + 1];
            //auto index2 = indexData[i + 2];
            triangles.push_back(triangle);
        }
    }
}

Scene::Scene(const char* filename)
{
    tinygltf::TinyGLTF loader;
    std::string err;
    std::string warn;
    //bool success = loader.LoadASCIIFromFile(&model, &err, &warn, filename);
    bool success = loader.LoadBinaryFromFile(&model, &err, &warn, filename);
    if (!warn.empty()) {
        std::cout << "Warn: " << warn << std::endl;
    }
    if (!err.empty()) {
        std::cerr << "Error: " << err << std::endl;
        assert(0);
    }

    if (!success) {
        std::cerr << "Failed to load glTF model." << std::endl;
        assert(0);
    }

    initTriangles();
}

void Scene::applyNodeTransform(const tinygltf::Node & node, glm::mat4x4& parentTransform)
{
    glm::mat4 localTransform(1.0f);
    glm::mat4 T(1.0f), R(1.0f), S(1.0f);

    if (!node.translation.empty()) {
        const auto& translation = node.translation;
        T = glm::translate(localTransform, glm::vec3(translation[0], translation[1], translation[2]));
    }

    if (!node.rotation.empty()) {
        const auto& rotation = node.rotation;
        // Please note: 
        // tinygltf stores quaternions in the order w, x, y, z
        // glm::mat constructor in the order x, y, z, w (It seems that glm quat module is not even internally consistent...)
        // Ref: https://stackoverflow.com/questions/48348509/glmquat-why-the-order-of-x-y-z-w-components-are-mixed
        glm::quat rotationQuaternion(rotation[3], rotation[0], rotation[1], rotation[2]);
        R = glm::mat4_cast(rotationQuaternion);
    }

    if (!node.scale.empty()) {
        const auto& scale = node.scale;
        S = glm::scale(localTransform, glm::vec3(scale[0], scale[1], scale[2]));
    }
    localTransform = T * R * S;
    // Update the parent transformation matrix with the node's transformation
    parentTransform = parentTransform * localTransform;
}

void Scene::initTriangles()
{
    auto initTransform = glm::mat4x4(1.0f);
    for (auto node : model.scenes[0].nodes) {
        traverseNode(model, node, initTransform);
    }
    return;
}

__global__ void freePrimitive(Primitive** primitives, int size) {
    for (size_t i = 0; i < size; i++)
    {
        delete primitives[i];
    }
}

void Scene::freeBuffer()
{
    int blockSize = 128;
    dim3 loadPrimitiveBlock((triangles.size() + (blockSize - 1)) / blockSize);
    //freePrimitive << <loadPrimitiveBlock, blockSize >> > (dev_primitives, getPrimitiveSize());
    freePrimitive << <1, 1>> > (dev_primitives, getPrimitiveSize());
    //for (size_t i = 0; i < getPrimitiveSize(); i++)
    //{
    //    delete dev_primitives[i];
    //}
    hipFree(dev_primitives);
}
