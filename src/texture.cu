#include "texture.cuh"
#include "stb_image.h"
#include <iostream>
#include "glm/glm.hpp"
CudaTexture::CudaTexture()
{}

CudaTexture::~CudaTexture()
{
	DeloadTexture();
}

void CudaTexture::LoadTexture(const char* path)
{
	DeloadTexture();
	if (path != "") {
		int nrChannels;
		stbi_set_flip_vertically_on_load(true);
		std::cout << "load texture from: " << path << std::endl;
		unsigned char* data = stbi_load(path, &m_width, &m_height, &nrChannels, 0);
		if (data) {
			hipChannelFormatDesc channelDesc =
				hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
			hipMallocArray(&m_cuArray, &channelDesc, m_width, m_height);
			const size_t spitch = m_width * sizeof(glm::vec4);
			// Copy data located at address h_data in host memory to device memory
			hipMemcpy2DToArray(m_cuArray, 0, 0, data, spitch, m_width * sizeof(glm::vec4), m_height, hipMemcpyHostToDevice);

			hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = m_cuArray;

			hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.addressMode[0] = hipAddressModeBorder;
			texDesc.addressMode[1] = hipAddressModeBorder;
			texDesc.filterMode = hipFilterModeLinear;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = 0;

			hipCreateTextureObject(&m_texObj, &resDesc, &texDesc, NULL);
		}
		else {
			std::cout << "Failed to load texture" << std::endl;
		}
		stbi_image_free(data);
	}
	else {
		std::cout << "invalid image file" << std::endl;
	}
}


void CudaTexture::LoadTexture(const std::string& path) {
	LoadTexture(path.c_str());
}

void CudaTexture::DeloadTexture()
{
	if (m_texObj != 0) {
		hipDestroyTextureObject(m_texObj);
		m_texObj = 0;
	}
	if (m_cuArray != nullptr) {
		hipFreeArray(m_cuArray);
		m_cuArray = nullptr;
	}
}
