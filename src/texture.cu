#include "hip/hip_runtime.h"
#include "texture.cuh"
#include "stb_image.h"
#include <iostream>
#include "glm/glm.hpp"

CudaTexture::CudaTexture(const char* path)
{
	LoadTexture(path);
}

CudaTexture::CudaTexture(const std::string& path)
	:CudaTexture(path.c_str())
{}

CudaTexture::CudaTexture(CudaTexture && other)
	:m_cuArray(other.m_cuArray),m_height(other.m_height),m_width(other.m_width),m_texObj(other.m_texObj)
{
	std::cout << "cuda texture move constructor " << std::endl;
	other.m_cuArray = nullptr;
	other.m_texObj = 0;
}

CudaTexture::~CudaTexture()
{
	DeloadTexture();
}

void CudaTexture::LoadTexture(const char* path)
{
	if (path != "") {
		int nrChannels;
		stbi_set_flip_vertically_on_load(true);
		std::cout << "load texture from: " << path << std::endl;
		auto data = stbi_loadf(path, &m_width, &m_height, &nrChannels, 4);
		if (data) {
			hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
			hipMallocArray(&m_cuArray, &channelDesc, m_width, m_height);
			const size_t spitch = m_width * sizeof(float4);
			// Copy data located at address h_data in host memory to device memory
			hipMemcpy2DToArray(m_cuArray, 0, 0, data, spitch, spitch, m_height, hipMemcpyHostToDevice);

			hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			

			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = m_cuArray;

			hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.addressMode[0] = hipAddressModeWrap;
			texDesc.addressMode[1] = hipAddressModeWrap;
			texDesc.filterMode = hipFilterModeLinear;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = 1;

			hipCreateTextureObject(&m_texObj, &resDesc, &texDesc, NULL);
		}
		else {
			std::cout << "Failed to load texture" << std::endl;
		}
		stbi_image_free(data);
	}
	else {
		std::cout << "invalid image file" << std::endl;
	}
}

void CudaTexture::DeloadTexture()
{
	if (m_texObj != 0) {
		std::cout << "destroy cuda texture object" << std::endl;
		hipDestroyTextureObject(m_texObj);
		m_texObj = 0;
	}
	if (m_cuArray != nullptr) {
		std::cout << "destroy cuda texture array" << std::endl;
		hipFreeArray(m_cuArray);
		m_cuArray = nullptr;
	}
}
