#include "hip/hip_runtime.h"
#include "cudaTexture.h"

#include "stb_image.h"
#include <iostream>

Texture2D::Texture2D()
	:m_RawImg(nullptr), 
    m_Width(0), 
    m_Height(0),
    m_TexObj(0),
    m_TexArray(nullptr)
{
}

Texture2D::Texture2D(const std::string& img_path, bool flip_v)
    :Texture2D()
{
    Create(img_path, flip_v);
}

Texture2D::~Texture2D()
{
	Free();
}

void Texture2D::Create(const std::string& img_path, bool flip_v)
{
	Free();

	// read image
	stbi_set_flip_vertically_on_load(flip_v);
	m_RawImg = stbi_loadf(img_path.c_str(), &m_Width, &m_Height, nullptr, 4);
	assert(m_RawImg != nullptr);

	// create array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    hipMallocArray(&m_TexArray, &channelDesc, m_Width, m_Height);

    int spitch = m_Width * sizeof(float4);
    const int& width = spitch;
    // send image to device
    hipMemcpy2DToArray(m_TexArray, 0, 0, m_RawImg, 
                            spitch,
                            width,
                            m_Height, hipMemcpyHostToDevice);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = m_TexArray;

    // texture description
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    hipCreateTextureObject(&m_TexObj, &resDesc, &texDesc, nullptr);
}

void Texture2D::Free()
{
    std::cout << "free texture" << std::endl;
	if (!m_RawImg) stbi_image_free(m_RawImg);
	m_RawImg = nullptr;

	m_Width = m_Height = 0;

    if (m_TexObj != 0)
    {
        hipDestroyTextureObject(m_TexObj);
        m_TexObj = 0;
    }
    SafeCudaFreeArray(m_TexArray);
}

CudaTexture2D::CudaTexture2D(const hipTextureObject_t& tex_obj)
	:m_TexObj(tex_obj)
{
}