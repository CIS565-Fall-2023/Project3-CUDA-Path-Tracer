#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include "common.h"
#include "efficient.h"

#define BLOCK_SIZE 128
#define DOUBLE_BLOCK_SIZE 2 * BLOCK_SIZE
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))


#define TIME_COMPACT 1

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
 
        __global__ void kernUpSweep(int n, int d, int* odata) {
            size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * (2 << d);
 
            if (index >= n) return;

            odata[index + (1 << (d + 1)) - 1] += odata[index + (1 << d) - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* odata) {
            size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * (2 << d);

            if (index >= n) return;

            // preserve the left child value
            int temp = odata[index + (1 << d) - 1];
            // left child copies the parent value
            odata[index + (1 << d) - 1] = odata[index + (1 << (d + 1)) - 1];
            // right child addes the parent value and the preserved left child value
            odata[index + (1 << (d + 1)) - 1] += temp;
        }

         /**
          *  apply shared memory to scan each block
          */
        __global__ void kernBlockScan(int n, int* odata, const int* idata, int* blockSums) {
            extern __shared__ int temp[];

            int thid = threadIdx.x;
            size_t index = blockIdx.x * blockDim.x + thid;

            // Load input into shared memory with boundary checks
            temp[2 * thid] = (2 * index < n) ? idata[2 * index] : 0;
            temp[2 * thid + 1] = (2 * index + 1 < n) ? idata[2 * index + 1] : 0;
            __syncthreads();

            int offset = 1;
           
            // Up-sweep (reduce) phase
            for (int d = blockDim.x; d > 0; d >>= 1) {
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }

            // Clear last element
            if (thid == 0) {
                blockSums[blockIdx.x] = temp[2 * blockDim.x - 1];
                temp[2 * blockDim.x - 1] = 0;
            }

            // Down-sweep phase
            for (int d = 1; d < 2 * blockDim.x; d *= 2) {
                offset >>= 1;
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();

            // Write results to device memory with boundary checks
            if (2 * index < n) {
                odata[2 * index] = temp[2 * thid];
            }

            if (2 * index + 1 < n) {
                odata[2 * index + 1] = temp[2 * thid + 1];
            }
        }

        __global__ void kernAddScannedBlockSums(int n, int* odata, const int* blockSums) {
            size_t index = blockIdx.x * blockDim.x + threadIdx.x;
            int blockSum = (blockIdx.x > 0) ? blockSums[blockIdx.x - 1] : 0;

            if (2 * index < n) {
                odata[2 * index] += blockSum;
            }

            if (2 * index + 1 < n) {
                odata[2 * index + 1] += blockSum;
            }
        }

        /**
         * Performs prefix-sum (aks scan) on idata using the shared memory, storing the result into odata
         */
        void scanShared(int n, int* odata, const int* idata) {
            int* dev_in, * dev_out, * dev_blockSums;
            
            const int log2ceil = ilog2ceil(n);
            const int fullSize = 1 << log2ceil;

            int gridSize = (fullSize + DOUBLE_BLOCK_SIZE - 1) / (DOUBLE_BLOCK_SIZE);
            // printf("gridSize: %d\n", gridSize);

            // allocate gpu memory
            hipMalloc((void**)&dev_in, fullSize * sizeof(int));
            hipMemset(dev_in, 0, fullSize * sizeof(int));
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            hipMalloc((void**)&dev_out, n * sizeof(int));

            hipMalloc((void**)&dev_blockSums, gridSize * sizeof(int));
            checkCUDAErrorFn("malloc dev_blockSums failed!");

            timer().startGpuTimer();
            kernBlockScan << <gridSize, BLOCK_SIZE, DOUBLE_BLOCK_SIZE * sizeof(int) >> > (fullSize, dev_out, dev_in, dev_blockSums);
            checkCUDAErrorFn("blockScan failed!");

            int* blockSums = new int[gridSize];
            hipMemcpy(blockSums, dev_blockSums, gridSize * sizeof(int), hipMemcpyDeviceToHost);
   
            for (int i = 1; i < gridSize; ++i) {
                blockSums[i] += blockSums[i - 1];
            }

            hipMemcpy(dev_blockSums, blockSums, gridSize * sizeof(int), hipMemcpyHostToDevice);
            delete[] blockSums;

            kernAddScannedBlockSums << <gridSize, BLOCK_SIZE >> > (fullSize, dev_out, dev_blockSums);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_in);
            hipFree(dev_out);
            hipFree(dev_blockSums);
        }

        void scanHelper(int n, int log2ceil, int* dev_out) {
            // up sweep 
            for (int d = 0; d <= log2ceil - 1; ++d) {
                // Adjust the grid size based on the depth of the sweep
                int gridSize = (n / (2 << d) + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernUpSweep << <gridSize, BLOCK_SIZE >> > (n, d, dev_out);
                checkCUDAErrorFn("up sweep failed!");
            }

            // set the last value to 0
            hipMemset(dev_out + n - 1, 0, sizeof(int));
            checkCUDAErrorWithLine("set the last value to zero failed!");

            // down sweep
            for (int d = log2ceil - 1; d >= 0; --d) {
                // Adjust the grid size based on the depth of the sweep
                int gridSize = (n / (2 << d) + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernDownSweep << <gridSize, BLOCK_SIZE >> > (n, d, dev_out);
                checkCUDAErrorFn("down sweep failed");
            }

        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_out;

            const int log2ceil = ilog2ceil(n);
            const long int fullSize = 1 << log2ceil;

            hipMalloc((void**)&dev_out, fullSize * sizeof(int));
            hipMemset(dev_out, 0, fullSize * sizeof(int));
            hipMemcpy(dev_out, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            scanHelper(fullSize, log2ceil, dev_out);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_out);
        }


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            int* dev_in, * dev_out, * dev_bools, * dev_scan;

            int boolLastVal, scanLastVal;

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            hipMalloc((void**)&dev_in, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_in failed!");
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("copy idata to dev_in failed!");

            hipMalloc((void**)&dev_out, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_out failed!");

            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_bools failed!");

#if TIME_COMPACT
            const int log2ceil = ilog2ceil(n);
            const long int fullSize = 1 << log2ceil;

            hipMalloc((void**)&dev_scan, fullSize * sizeof(int));
            checkCUDAErrorFn("malloc dev_scan failed!");
            hipMemset(dev_scan, 0, n * sizeof(int));
#else
            hipMalloc((void**)&dev_scan, n * sizeof(int));
            checkCUDAErrorFn("malloc dev_scan failed!");
#endif

#if TIME_COMPACT
            timer().startGpuTimer();
#endif
            // map the bool array
            StreamCompaction::Common::kernMapToBoolean << <gridSize, BLOCK_SIZE >> > (n, dev_bools, dev_in);
            checkCUDAErrorFn("map bool array failed!");

            
#if TIME_COMPACT
            // scan the bool array
            hipMemcpy(dev_scan, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);

            // up sweep
            for (int d = 0; d <= log2ceil - 1; ++d) {
                int dynamicGridSize = (fullSize / (2 << d) + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernUpSweep << <dynamicGridSize, BLOCK_SIZE >> > (fullSize, d, dev_scan);
                checkCUDAErrorFn("up sweep failed!");
            }

            // set the last value to 0
            hipMemset(dev_scan + fullSize - 1, 0, sizeof(int));
            
            // down sweep
            for (int d = log2ceil - 1; d >= 0; --d) {
                int dynamicGridSize = (fullSize / (2 << d) + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernDownSweep << <dynamicGridSize, BLOCK_SIZE >> > (fullSize, d, dev_scan);
                checkCUDAErrorFn("down sweep failed");
            }
#else 
            // scan(n, dev_scan, dev_bools);
            scanShared(n, dev_scan, dev_bools);
#endif
            // scatter
            StreamCompaction::Common::kernScatter << <gridSize, BLOCK_SIZE >> > (n, dev_out, dev_in, dev_bools, dev_scan);
            checkCUDAErrorFn("scatter failed!");
#if TIME_COMPACT
            timer().endGpuTimer();
#endif
            // store the last value of the bool array
            hipMemcpy(&boolLastVal, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("copy last bool value to host failed!");

            // store the last value of the scan results
            hipMemcpy(&scanLastVal, dev_scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("copy last bool value to host failed!");

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("copy dev_out to odata failed!");

            // free memory
            hipFree(dev_in);
            hipFree(dev_out);
            hipFree(dev_bools);
            hipFree(dev_scan);

            return scanLastVal + boolLastVal;
        }
    }
}
