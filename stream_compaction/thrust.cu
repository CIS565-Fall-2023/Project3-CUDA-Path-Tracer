#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata -> dev_idata failed!");

            timer().startGpuTimer();

            thrust::device_ptr<int> dv_in(dev_idata);
            thrust::device_ptr<int> dv_out(dev_odata);
            thrust::exclusive_scan(dv_in, dv_in + n, dv_out);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata -> odata failed!");

            hipFree(dev_idata);
            hipFree(dev_odata);
        }

        void sort(int n, int* odata, const int* idata) {
            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata -> dev_idata failed!");

            timer().startGpuTimer();

            thrust::device_ptr<int> dv_in(dev_idata);
            thrust::sort(thrust::device, dev_idata, dev_idata + n);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata -> odata failed!");
            hipFree(dev_idata);
        }
    }
}
